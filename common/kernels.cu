#include "hip/hip_runtime.h"
/*
 * author: ck
 * created: 08.12.2011
 * advisor: atc
 */


#include "settings.h"

#include "cuPrintf.cu"



// generates pair-wise multiplication result
__global__ void hadamard_mul(double* d_A, double* d_B, double* d_C, size_t C_element_number, bool print, int to_power_A, int to_power_B){
  size_t tid = blockIdx.x * blockDim.x + threadIdx.x;
  if (tid < C_element_number){

    d_C[tid] = pow(d_A[tid], to_power_A) * pow(d_B[tid], to_power_B);

    if( print ){
      double result = pow(d_A[tid], to_power_A) * pow(d_B[tid], to_power_B);
      cuPrintf("hadamard_mul result %f \n", result);
    }

  }
}

// generates pair-wise division result
__global__ void hadamard_div(double* d_A, double* d_B, double* d_C, size_t C_element_number, bool print, int to_power_A, int to_power_B){
  size_t tid = blockIdx.x * blockDim.x + threadIdx.x;
  if (tid < C_element_number){
    d_C[tid] = pow(d_A[tid], to_power_A) / pow(d_B[tid], to_power_B);

    if( print ){
      double result = pow(d_A[tid], to_power_A) / pow(d_B[tid], to_power_B);
      cuPrintf("hadamard_div result %f \n", result);
    }

  }
}

// generates pair-wise summation result
__global__ void hadamard_sum(double* d_A, double* d_B, double* d_C, size_t C_element_number, bool print, int to_power_A, int to_power_B){
  size_t tid = blockIdx.x * blockDim.x + threadIdx.x;
  if (tid < C_element_number){
    d_C[tid] = pow(d_A[tid], to_power_A) + pow(d_B[tid], to_power_B);

    if( print ){
      double result = pow(d_A[tid], to_power_A) + pow(d_B[tid], to_power_B);
      cuPrintf("hadamard_sum result %f \n", result);
    }

  }
}


// generates the full result tensor
__global__ void genFullResult(size_t* d_total_cards, size_t ndims,
                              size_t* d_strides_A, size_t* d_strides_B, size_t* d_strides_F,
                              double* d_A, double* d_B, double* d_F,
                              size_t F_element_number, size_t A_element_number, size_t B_element_number,
                              size_t use_multiplication,
                              bool print,
                              int to_power_A, int to_power_B){

  size_t tid = blockIdx.x * blockDim.x + threadIdx.x;
  size_t d_inds_F;// = (size_t*) malloc(sizeof(size_t)*ndims);

  if (tid < F_element_number){

    // for each element of the full result tensor
    //      multiply corresponding elements of input tensors A and B

    if( print ){
      cuPrintf("tid %d element numbers F %d A %d B %d \n",tid,F_element_number, A_element_number, B_element_number);
    }

    size_t F_ind=0;
    size_t A_ind=0;
    size_t B_ind=0;
    for ( size_t dim=ndims-1; ; dim--){

      if ( tid / d_strides_F[dim] > 0 ){
        d_inds_F = tid / d_strides_F[dim];
        tid -= d_inds_F*d_strides_F[dim];
      }else{
        d_inds_F = 0;
      }

      F_ind += d_strides_F[dim] * d_inds_F;
      A_ind += d_strides_A[dim] * d_inds_F;
      B_ind += d_strides_B[dim] * d_inds_F;


      if( print ){
        size_t tmp= d_strides_F[dim];
        cuPrintf("F_ind %d d_strides_F %d d_inds_F %d\n", F_ind, F_ind, tmp);
      }

      if(dim == 0) break;
    }


    if ( A_ind >= A_element_number ){
      if( print ){
        cuPrintf("A preventing index overflow index %d max %d\n",A_ind, A_element_number-1);
      }
      A_ind = A_element_number-1;
    }

    if ( B_ind >= B_element_number ){
      if( print ){
        cuPrintf("B preventing index overflow index %d max %d\n",B_ind, B_element_number-1);
      }
      B_ind = B_element_number-1;
    }


    if (use_multiplication == 1)
      d_F[F_ind] = pow(d_A[A_ind], to_power_A) * pow(d_B[B_ind], to_power_B);
    else
      d_F[F_ind] = pow(d_A[A_ind], to_power_A) / pow(d_B[B_ind], to_power_B);


    if( print ){
      double tmpval = 0;

      if (use_multiplication == 1)
        tmpval = pow(d_A[A_ind], to_power_A) * pow(d_B[B_ind], to_power_B);
      else
        tmpval = pow(d_A[A_ind], to_power_A) / pow(d_B[B_ind], to_power_B);

      double Aval = d_A[A_ind];
      double Bval = d_B[B_ind];

      cuPrintf("tidABC %d: d_F[%d] = d_A[%d] * d_B[%d] = %f op %f = %f \n", tid, F_ind, A_ind, B_ind, Aval, Bval, tmpval);
    }

  }
}


// for each element of d_C (tid corresponds to a single iteration)
//    loop over every zero cardinality dimension summing in tmp_sum
//    store tmp_sum as corresponding element of d_C
__global__ void contractFintoC(size_t ndims,
                               size_t* d_strides_F, size_t* d_strides_C,
                               double* d_F, double* d_C,
                               size_t C_element_number,
                               size_t* d_zero_cardinality_dim_tuples_C,
                               size_t zero_cardinality_dim_tuple_size_C,
                               size_t zero_cardinality_dim_tuples_C_element_number,
                               bool print) {

  size_t tid = blockIdx.x * blockDim.x + threadIdx.x;

  size_t d_inds_C[20]; // 20 dimensions limit


  if ( tid < C_element_number ){
    // calculate index for this tid
    size_t C_ind=0;
    for ( size_t dim=ndims-1; ; dim--){
      if (d_strides_C[dim] != 0){
        if ( tid / d_strides_C[dim] > 0 ){
          d_inds_C[dim] = tid / d_strides_C[dim];
          tid -= d_inds_C[dim]*d_strides_C[dim];
        }else{
          d_inds_C[dim] = 0;
        }
      }


      C_ind += d_strides_C[dim] * d_inds_C[dim];

      // size_t tmp= d_strides_C[dim];
      // size_t tmp1= d_inds_C[dim];
      // cuPrintf("dim %d C_ind %d d_strides_C %d d_inds_C %d\n",dim, C_ind, tmp, tmp1);


      if(dim == 0) break;
    }

    // for(size_t i=0; i<ndims; i++){
    //  size_t tmp=d_inds_C[i];
    //  cuPrintf("d_inds_C %d\n",tmp);
    //   }


    // calculate contraction value for this index of output tensor C
    double tmp_sum=0;

    // d_zero_cardinality_dim_tuples_C contains tuples of size zero_cardinality_dim_tuple_size_C
    // these correspond to the set of all possible indices over zero cardinality indices of tensor C

    if( print ){
      cuPrintf("cont: zero_cardinality_dim_tuples_C_element_number %d\n",zero_cardinality_dim_tuples_C_element_number);
      cuPrintf("cont: zero_cardinality_dim_tuple_size_C %d\n",zero_cardinality_dim_tuple_size_C);
    }

    for ( size_t iter=0;
          iter < zero_cardinality_dim_tuples_C_element_number; ){

      size_t F_ind = 0;
      for ( size_t dim=0 ; dim<ndims; dim++){
        if ( d_strides_F[dim] == 0 ){
          continue;
        }

        if ( d_strides_C[dim] == 0 ){
          F_ind += d_strides_F[dim] * d_zero_cardinality_dim_tuples_C[iter];

          if( print ){
            size_t tmp = d_strides_F[dim] * d_zero_cardinality_dim_tuples_C[iter];
            size_t tmp1 = d_strides_F[dim];
            size_t tmp2 = d_zero_cardinality_dim_tuples_C[iter];
            cuPrintf("cont: F_ind val %d, stride %d, inds %d\n",tmp, tmp1, tmp2 );
          }

          iter++;
        }else{

          F_ind += d_strides_F[dim] * d_inds_C[dim];

          if( print ){
            size_t tmp = d_strides_F[dim] * d_inds_C[dim];
            size_t tmp1 = d_strides_F[dim];
            size_t tmp2 = d_inds_C[dim];
            cuPrintf("cont: F_ind else val %d, stride %d, inds %d\n",tmp, tmp1, tmp2 );
          }
        }
      }

      if( print ){
        double kek=d_F[F_ind];
        cuPrintf("cont: F_ind %d d_F[F_ind] %f\n", F_ind, kek);
      }
      tmp_sum += d_F[F_ind];
    }





    // store this element of d_C
    if( print ){
      cuPrintf("cont: store C_ind %d C_element_number %d value %f\n",C_ind, C_element_number, tmp_sum);
    }
    d_C[C_ind] = tmp_sum;
  }
}







// for each element of d_C (tid corresponds to a single iteration)
//    loop over every zero cardinality dimension summing in tmp_sum
//    store tmp_sum as corresponding element of d_C
__global__ void calculate_C_mops(size_t ndims,
                                 size_t operand_num,

                                 size_t** d_strides_operand_pointers, //

                                 size_t* d_strides_output,
                                 size_t* d_cards_F,

                                 size_t** d_cards_operand_pointers, //
                                 double** d_operand_pointers,       //

                                 double* d_output,
                                 //size_t* operand_element_numbers,
                                 size_t output_element_number,
                                 size_t use_multiplication,
                                 bool print
                                 //int* to_power_operands             // -
                                 ){
  size_t tid = blockIdx.x * blockDim.x + threadIdx.x;

  size_t d_inds_output[20]; // 20 dimensions limit

  if ( tid < output_element_number ){

    ///// calculate output index for this tid
    size_t output_ind=0;
    for ( size_t dim=ndims-1; ; dim--){
      if (d_strides_output[dim] != 0){
        if ( tid / d_strides_output[dim] > 0 ){
          d_inds_output[dim] = tid / d_strides_output[dim];
          tid -= d_inds_output[dim]*d_strides_output[dim];
        }else{
          d_inds_output[dim] = 0;
        }
      }

      output_ind += d_strides_output[dim] * d_inds_output[dim];

      // size_t tmp= d_strides_C[dim];
      // size_t tmp1= d_inds_C[dim];
      // cuPrintf("dim %d C_ind %d d_strides_C %d d_inds_C %d\n",dim, C_ind, tmp, tmp1);

      if(dim == 0) break;
    }

    if ( print ){
      // for(size_t i=0; i<ndims; i++){
      //   size_t tmp=d_inds_output[i];
      //   cuPrintf("d_inds_output dim %d : %d \n", i, tmp);
      // }
      cuPrintf("OUTPUT IND %d\n",output_ind);
    }

    /////////////////////////////////////////////

    // new value for d_output[output_ind]
    double val=0;

    double tmp_vals[2] = {1, 1};
    size_t tmp_val_ind = 0;


    // for each operand
    for( size_t operand=0; operand<operand_num; operand++){

      ///// find operand index(s) for current output_ind

      size_t operand_ind = 0;
      size_t tmp_output_ind = output_ind;
      size_t tmp_digit;

      for ( size_t dim=ndims-1; ; dim--){
        if (d_strides_output[dim] != 0){
          if ( tmp_output_ind / d_strides_output[dim] > 0 ){
            tmp_digit = tmp_output_ind / d_strides_output[dim];
            tmp_output_ind -= tmp_digit*d_strides_output[dim];
          }else{
            tmp_digit = 0;
          }
        }else{
	  tmp_digit = 0;
	}

        operand_ind += d_strides_operand_pointers[operand][dim] * tmp_digit;

        if( print ){
	  size_t tmp = d_strides_operand_pointers[operand][dim];
	  cuPrintf("d_strides_operand_pointers[%d][%d] %d tmp_digit %d -> operand_ind %d\n",operand, dim, tmp, tmp_digit, operand_ind); 
        }

        if(dim == 0) break;
      }


      // if ( operand_ind >= operand_element_numbers[operand] ){
      //   operand_ind = operand_element_numbers[operand]-1;
      //   if( print ){
      //     cuPrintf("preventing operand index overflow index %d max %d\n",operand_ind, operand_element_numbers[operand]-1);
      //   }
      // }
      /////////////////////////////////////////////


      ///// increment val for this operand


      // cardinalities of contraction indices
      //size_t d_contraction_cards[20]; // 20 dimensions limit
      // number of indices to contract
      size_t contraction_index_num = 0;

      // contains contraction indices of the operand
      size_t d_contraction_ind_operand[20] = {0}; // 20 dimension limit 
      size_t dcio_i=0;
      for( size_t dim=0; dim<ndims; dim++){
        if ( d_strides_output[dim] == 0 && d_cards_operand_pointers[operand][dim] != 0 ){
          //d_contraction_cards[contraction_index_num] = d_cards_F[dim];
          contraction_index_num++;

	  // store which index of the operand we contract
	  d_contraction_ind_operand[dcio_i]=dim;
	  dcio_i++;
        }

	if(print){
	  size_t tmp=d_strides_output[dim];
	  size_t tmp1= d_cards_operand_pointers[operand][dim];
	  cuPrintf("operand %d d_strides_output[%d] %d d_cards_operand_pointers[%d][%d] %d contraction_index_num %d\n", operand, dim, tmp, operand, dim, tmp1, contraction_index_num);
	}
      }

      // if(print){
      // 	if ( blockIdx.x * blockDim.x + threadIdx.x == 0) {
      // 	  size_t elnum=1;
      // 	  for(int d=0; d<ndims; d++)
      // 	    if (d_cards_operand_pointers[operand][d] != 0)
      // 		elnum *= d_cards_operand_pointers[operand][d];

      // 	  cuPrintf("operand %d elnum %d\n" , operand, elnum);
      // 	  for (int i=0; i<elnum; i++){
      // 	    double tmp1=d_operand_pointers[operand][ i ];
      // 	    cuPrintf("operand %d d_operand_pointers[%d][%d] = %f \n",operand, operand, i, tmp1);
      // 	  }
      // 	}
      // }


      if (print){
      	// print d_output
      	if ( blockIdx.x * blockDim.x + threadIdx.x == 0){
      	  for(int i=0; i<60; i++){
      	    double tmp=d_output[i];
      	    cuPrintf("d_output[%d] = %f\n", i, tmp);
      	  }
      	}
      }


      // // calculate displacement on d_operands due to previous operands
      // size_t prev_operand_element_num=0;
      // for( size_t prev_operands=0; prev_operands<operand; prev_operands++){
      //   prev_operand_element_num += operand_element_numbers[prev_operands];
      // }

      // if V_output == V_operand
      //   operand and output indices are the same only multiply
      if( contraction_index_num == 0 ){

        //val *= pow(d_operand_pointers[opreand][ operand_ind ], to_power_operands[operand]);
	if (val == 0){
	  val = d_operand_pointers[operand][ operand_ind ];
	}else{
	  val *= d_operand_pointers[operand][ operand_ind ];
	}


	d_output[output_ind] += val;
	cuPrintf("d_output ASSIGN output_ind %d = val %d \n",output_ind, val);


	if(print && blockIdx.x * blockDim.x + threadIdx.x == 0){
	  cuPrintf("contraction_index_num=0 val %f d_operand_pointers[%d][%d] = %f ", val, operand, operand_ind);
	}
	
      }else{
        // operand and output indices are not the same, must multiply and contract

	double prev_val;
	if( val == 0 ){
	  prev_val = 1;
	}else{
	  prev_val = val;
	}

        // val += prev_val * operand[ base_index + stride_contraction_index ]

        // for each combination of the contraction indices, perform val += operation
        bool not_done=true;
        size_t d_contraction_ind[20] = {0}; // 20 dimensions limit

        do{

          // d_contraction_cards contains cardinalities of indices to be contracted on operand
          // careful of indices which have zero cardinality both in output and operand -> NOT REQUIRED ONLY CALCULATING STRIDES!


          // use this configuration of d_contraction_indices and increment val

          size_t contraction_stride = 0;
	  size_t i=0;
          for( size_t d=0; d<ndims; d++){
	    if ( d_strides_output[d] == 0 && d_cards_operand_pointers[operand][d] != 0 ){
	      contraction_stride += d_strides_operand_pointers[operand][d] * d_contraction_ind[i];
	      i++;
	      cuPrintf("contraction_stride += %d * %d = %d\n", d_strides_operand_pointers[operand][d] , d_contraction_ind[i], contraction_stride);
	    }
          }

          //val += prev_val * pow(d_operand_pointers[operand][operand_ind + contraction_stride ], to_power_operands[operand]);
          tmp_vals[tmp_val_ind] *= prev_val * d_operand_pointers[operand][operand_ind + contraction_stride ];

	  if( tmp_val_ind == 0 ) tmp_val_ind=1;
	  else tmp_val_ind=0;
	  

	  if ( print 
	       //&& blockIdx.x * blockDim.x + threadIdx.x == 0 
	       ){
	    double tmp = d_operand_pointers[operand][operand_ind + contraction_stride ];
	    cuPrintf("val increment output_ind %d = val %f prev_val %f d_operand_pointers[%d][%d +%d] %f\n",output_ind, val, prev_val, operand, operand_ind, contraction_stride, tmp);
	    for(int i=0; i<contraction_index_num; i++){
	      size_t tmp = d_contraction_ind[i];
	      cuPrintf("cin: operand %d cin[%i] = %d \n", operand, i, tmp);
	    }
	  }


          // increment d_contraction_ind
          for (size_t dim=0; dim<contraction_index_num; dim++){
            // if we have NOT reached limit of this dimension
            if( d_contraction_ind[dim] != (d_cards_operand_pointers[operand][d_contraction_ind_operand[dim]]-1) && d_cards_operand_pointers[operand][d_contraction_ind_operand[dim]] != 0 ){
              // increment this dimension
              d_contraction_ind[dim]++;
              break;
            }else{
              // we have reached limit of this dimension

              // if next dimension is at limit as well, skip this dimension, operation will take place in next dimension
              if( dim != (contraction_index_num-1) &&
                  (d_contraction_ind[dim+1] == (d_cards_operand_pointers[operand][d_contraction_ind_operand[dim+1]]-1) || d_cards_operand_pointers[operand][d_contraction_ind_operand[dim+1]] == 0 ) ){
                //std::cout << "skip" << std::endl;
                continue;
              }else{

                // if this is the last dimension (and it is full) no increment is possible increment error
                if (dim == contraction_index_num-1){
                  //h_ct->increment_error = 1;
                  not_done = false;
                  break;
                }

                // make this and all previous dimensions zero
                for (int dim_prev=dim; dim_prev>=0 ; dim_prev--){
                  d_contraction_ind[dim_prev] = 0;
                }
                // increment next dimension
                d_contraction_ind[dim+1]++;
                break;
              }
            }
          }
        }while( not_done );
      }
    }
    d_output[output_ind] += tmp_vals[0] + tmp_vals[1];
    cuPrintf("d_output ASSIGN output_ind %d = val %f \n",output_ind, d_output[output_ind]);
  }
}


__global__ void printData(double* data, size_t count, size_t id){
  cuPrintf("printData id %d", id);
  for(int i=0; i<6; i++){
    double tmp=data[i];
    cuPrintf("data[%d] = %e\n", i, tmp);
  }
}

