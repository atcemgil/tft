#include "hip/hip_runtime.h"
/*
 * author: ck
 * created: 08.12.2011
 * advisor: atc
 */


#include "settings.h"

#include "cuPrintf.cu"



// generates pair-wise multiplication result
__global__ void hadamard_mul(double* d_A, double* d_B, double* d_C, size_t C_element_number, bool print, int to_power_A, int to_power_B){
  size_t tid = blockIdx.x * blockDim.x + threadIdx.x;
  if (tid < C_element_number){

    d_C[tid] = pow(d_A[tid], to_power_A) * pow(d_B[tid], to_power_B);

    if( print ){
      double result = pow(d_A[tid], to_power_A) * pow(d_B[tid], to_power_B);
      cuPrintf("hadamard_mul result %f \n", result);
    }

  }
}

// generates pair-wise division result
__global__ void hadamard_div(double* d_A, double* d_B, double* d_C, size_t C_element_number, bool print, int to_power_A, int to_power_B){
  size_t tid = blockIdx.x * blockDim.x + threadIdx.x;
  if (tid < C_element_number){
    d_C[tid] = pow(d_A[tid], to_power_A) / pow(d_B[tid], to_power_B);

    if( print ){
      double result = pow(d_A[tid], to_power_A) / pow(d_B[tid], to_power_B);
      cuPrintf("hadamard_div result %f \n", result);
    }

  }
}

// generates pair-wise summation result
__global__ void hadamard_sum(double* d_A, double* d_B, double* d_C, size_t C_element_number, bool print, int to_power_A, int to_power_B){
  size_t tid = blockIdx.x * blockDim.x + threadIdx.x;
  if (tid < C_element_number){
    d_C[tid] = pow(d_A[tid], to_power_A) + pow(d_B[tid], to_power_B);

    if( print ){
      double result = pow(d_A[tid], to_power_A) + pow(d_B[tid], to_power_B);
      cuPrintf("hadamard_sum result %f \n", result);
    }

  }
}


// generates the full result tensor
__global__ void genFullResult(size_t* d_total_cards, size_t ndims,
                              size_t* d_strides_A, size_t* d_strides_B, size_t* d_strides_F,
                              double* d_A, double* d_B, double* d_F,
                              size_t F_element_number, size_t A_element_number, size_t B_element_number,
                              size_t use_multiplication,
                              bool print,
                              int to_power_A, int to_power_B){

  size_t tid = blockIdx.x * blockDim.x + threadIdx.x;
  size_t d_inds_F;// = (size_t*) malloc(sizeof(size_t)*ndims);

  if (tid < F_element_number){

    // for each element of the full result tensor
    //      multiply corresponding elements of input tensors A and B

    if( print ){
      cuPrintf("tid %d element numbers F %d A %d B %d \n",tid,F_element_number, A_element_number, B_element_number);
    }

    size_t F_ind=0;
    size_t A_ind=0;
    size_t B_ind=0;
    for ( size_t dim=ndims-1; ; dim--){

      if ( tid / d_strides_F[dim] > 0 ){
        d_inds_F = tid / d_strides_F[dim];
        tid -= d_inds_F*d_strides_F[dim];
      }else{
        d_inds_F = 0;
      }

      F_ind += d_strides_F[dim] * d_inds_F;
      A_ind += d_strides_A[dim] * d_inds_F;
      B_ind += d_strides_B[dim] * d_inds_F;


      if( print ){
        size_t tmp= d_strides_F[dim];
        cuPrintf("F_ind %d d_strides_F %d d_inds_F %d\n", F_ind, F_ind, tmp);
      }

      if(dim == 0) break;
    }


    if ( A_ind >= A_element_number ){
      if( print ){
        cuPrintf("A preventing index overflow index %d max %d\n",A_ind, A_element_number-1);
      }
      A_ind = A_element_number-1;
    }

    if ( B_ind >= B_element_number ){
      if( print ){
        cuPrintf("B preventing index overflow index %d max %d\n",B_ind, B_element_number-1);
      }
      B_ind = B_element_number-1;
    }


    if (use_multiplication == 1)
      d_F[F_ind] = pow(d_A[A_ind], to_power_A) * pow(d_B[B_ind], to_power_B);
    else
      d_F[F_ind] = pow(d_A[A_ind], to_power_A) / pow(d_B[B_ind], to_power_B);


    if( print ){
      double tmpval = 0;

      if (use_multiplication == 1)
        tmpval = pow(d_A[A_ind], to_power_A) * pow(d_B[B_ind], to_power_B);
      else
        tmpval = pow(d_A[A_ind], to_power_A) / pow(d_B[B_ind], to_power_B);

      double Aval = d_A[A_ind];
      double Bval = d_B[B_ind];

      cuPrintf("tidABC %d: d_F[%d] = d_A[%d] * d_B[%d] = %f op %f = %f \n", tid, F_ind, A_ind, B_ind, Aval, Bval, tmpval);
    }

  }
}


// for each element of d_C (tid corresponds to a single iteration)
//    loop over every zero cardinality dimension summing in tmp_sum
//    store tmp_sum as corresponding element of d_C
__global__ void contractFintoC(size_t ndims,
                               size_t* d_strides_F, size_t* d_strides_C,
                               double* d_F, double* d_C,
                               size_t C_element_number,
                               size_t* d_zero_cardinality_dim_tuples_C,
                               size_t zero_cardinality_dim_tuple_size_C,
                               size_t zero_cardinality_dim_tuples_C_element_number,
                               bool print) {

  size_t tid = blockIdx.x * blockDim.x + threadIdx.x;

  size_t d_inds_C[20]; // 20 dimensions limit


  if ( tid < C_element_number ){
    // calculate index for this tid
    size_t C_ind=0;
    for ( size_t dim=ndims-1; ; dim--){
      if (d_strides_C[dim] != 0){
        if ( tid / d_strides_C[dim] > 0 ){
          d_inds_C[dim] = tid / d_strides_C[dim];
          tid -= d_inds_C[dim]*d_strides_C[dim];
        }else{
          d_inds_C[dim] = 0;
        }
      }


      C_ind += d_strides_C[dim] * d_inds_C[dim];

      // size_t tmp= d_strides_C[dim];
      // size_t tmp1= d_inds_C[dim];
      // cuPrintf("dim %d C_ind %d d_strides_C %d d_inds_C %d\n",dim, C_ind, tmp, tmp1);


      if(dim == 0) break;
    }

    // for(size_t i=0; i<ndims; i++){
    //  size_t tmp=d_inds_C[i];
    //  cuPrintf("d_inds_C %d\n",tmp);
    //   }


    // calculate contraction value for this index of output tensor C
    double tmp_sum=0;

    // d_zero_cardinality_dim_tuples_C contains tuples of size zero_cardinality_dim_tuple_size_C
    // these correspond to the set of all possible indices over zero cardinality indices of tensor C

    if( print ){
      cuPrintf("cont: zero_cardinality_dim_tuples_C_element_number %d\n",zero_cardinality_dim_tuples_C_element_number);
      cuPrintf("cont: zero_cardinality_dim_tuple_size_C %d\n",zero_cardinality_dim_tuple_size_C);
    }

    for ( size_t iter=0;
          iter < zero_cardinality_dim_tuples_C_element_number; ){

      size_t F_ind = 0;
      for ( size_t dim=0 ; dim<ndims; dim++){
        if ( d_strides_F[dim] == 0 ){
          continue;
        }

        if ( d_strides_C[dim] == 0 ){
          F_ind += d_strides_F[dim] * d_zero_cardinality_dim_tuples_C[iter];

          if( print ){
            size_t tmp = d_strides_F[dim] * d_zero_cardinality_dim_tuples_C[iter];
            size_t tmp1 = d_strides_F[dim];
            size_t tmp2 = d_zero_cardinality_dim_tuples_C[iter];
            cuPrintf("cont: F_ind val %d, stride %d, inds %d\n",tmp, tmp1, tmp2 );
          }

          iter++;
        }else{

          F_ind += d_strides_F[dim] * d_inds_C[dim];

          if( print ){
            size_t tmp = d_strides_F[dim] * d_inds_C[dim];
            size_t tmp1 = d_strides_F[dim];
            size_t tmp2 = d_inds_C[dim];
            cuPrintf("cont: F_ind else val %d, stride %d, inds %d\n",tmp, tmp1, tmp2 );
          }
        }
      }

      if( print ){
        double kek=d_F[F_ind];
        cuPrintf("cont: F_ind %d d_F[F_ind] %f\n", F_ind, kek);
      }
      tmp_sum += d_F[F_ind];
    }





    // store this element of d_C
    if( print ){
      cuPrintf("cont: store C_ind %d C_element_number %d value %f\n",C_ind, C_element_number, tmp_sum);
    }
    d_C[C_ind] = tmp_sum;
  }
}







// for each element of d_C (tid corresponds to a single iteration)
//    loop over every zero cardinality dimension summing in tmp_sum
//    store tmp_sum as corresponding element of d_C
__global__ void calculate_C_mops(size_t ndims,
                                 size_t operand_num,

                                 size_t** d_strides_operand_pointers, //

                                 size_t* d_strides_output,
                                 size_t* d_cards_F,

                                 size_t** d_cards_operand_pointers, //
                                 double** d_operand_pointers,       //

                                 double* d_output,
                                 //size_t* operand_element_numbers,
                                 size_t output_element_number,
                                 size_t use_multiplication,
                                 bool print,
                                 size_t opnum
                                 //int* to_power_operands             // -
                                 ){
  size_t tid = blockIdx.x * blockDim.x + threadIdx.x;

  size_t d_inds_output[20] = {0}; // 20 dimensions limit

  if ( tid < output_element_number ){

    cuPrintf("selam %d\n",opnum);

    for( size_t i=0; i<ndims; i++ ){
      cuPrintf("calculate_C_mops d_strides_output[%d] = %d , output element number %d\n", i, d_strides_output[i], output_element_number);
    }

    ///// calculate output index for this tid
    size_t output_ind=0;
    for ( size_t dim=ndims-1; ; dim--){
      if (d_strides_output[dim] != 0){
        if ( tid / d_strides_output[dim] > 0 ){
          d_inds_output[dim] = tid / d_strides_output[dim];
          tid -= d_inds_output[dim]*d_strides_output[dim];
        }else{
          d_inds_output[dim] = 0;
        }
      }

      output_ind += d_strides_output[dim] * d_inds_output[dim];

      if(dim == 0) break;
    }

    if ( print ){
      for(size_t i=0; i<ndims; i++){
        size_t tmp=d_inds_output[i];
        cuPrintf("d_inds_output dim %d : %d \n", i, tmp);
      }
      cuPrintf("OUTPUT IND %d\n",output_ind);
    }

    d_output[output_ind]=0;




    /////////////////////////////////////////////





    // for all contraction indices

    // V_output \ V_F
    size_t d_contraction_ind[20] = {0}; // 20 dimensions limit
    size_t d_contraction_dim_cards[20] = {0}; // 20 dimensions limit
    size_t contraction_index_num=0;
    for( size_t dim=0; dim<ndims; dim++){
      if ( d_strides_output[dim] == 0 && d_cards_F[dim] != 0 ){
	d_contraction_dim_cards[contraction_index_num]=d_cards_F[dim];
        contraction_index_num++;
      }
      if(print){
        cuPrintf("contraction_index_num %d\n", contraction_index_num);
      }
    }



    bool not_done = true;

    do{
      // for each contraction index find a multiplication of operands and sum them
      double val=1;

      // get elements from each operand using d_contraction_ind and multiply
      for( size_t operand=0; operand<operand_num; operand++){


	// find operand index
	// use d_inds_output
	// if d_inds_output is zero use d_contraction_ind
	size_t op_inds=0;
	size_t contr_ind = 0;
	for( size_t d=0; d<ndims; d++){
	  if (d_strides_operand_pointers[operand][d] != 0){

	    //cuPrintf("d_inds_output[%d] = %d\n", d, d_inds_output[d]);

	    if ( d_strides_output[d] == 0 ){
	      op_inds += d_contraction_ind[contr_ind] * d_strides_operand_pointers[operand][d];
	      //cuPrintf("(contr_ind %d) op_inds += %d * %d (dim %d) = %d\n", contr_ind, d_contraction_ind[contr_ind], d_strides_operand_pointers[operand][d], d, op_inds);
	      contr_ind++;
	    }else{
	      op_inds += d_inds_output[d] * d_strides_operand_pointers[operand][d];
	      //cuPrintf("op_inds += %d * %d (dim %d) = %d\n", d_inds_output[d], d_strides_operand_pointers[operand][d], d);
	    }
	  }
	}

        val *= d_operand_pointers[operand][ op_inds ];
	//cuPrintf("val increment operand %d op_inds %d d_operand_pointers %f new val %f\n", operand, op_inds, d_operand_pointers[operand][ op_inds ], val);
      }


      d_output[output_ind] += val;
      cuPrintf("d_output increment output_ind %d val %f new d_output %f\n", output_ind, val, d_output[output_ind]);


      for(size_t i=0; i<contraction_index_num; i++){
	//cuPrintf("d_contraction_ind[%d] = %d\n", i, d_contraction_ind[i]);
      }

      // increment d_contraction_ind for next loop OR end iteration if done
      for (size_t dim=0; dim<contraction_index_num; dim++){
        // if we have NOT reached limit of this dimension
        if( d_contraction_ind[dim] != (d_contraction_dim_cards[dim]-1) ){
          // increment this dimension
	  //cuPrintf("INCREMENT %d %d\n",d_contraction_ind[dim], (d_contraction_dim_cards[dim]-1));
          d_contraction_ind[dim]++;
          break;
        }else{
          // we have reached limit of this dimension

          // if next dimension is at limit as well, skip this dimension, operation will take place in next dimension
          if( dim != (contraction_index_num-1) &&
              (d_contraction_ind[dim+1] == (d_contraction_dim_cards[dim+1]-1) || d_contraction_dim_cards[dim+1] == 0 ) ){
	    //cuPrintf("SKIP\n");
            continue;
          }else{

            // if this is the last dimension (and it is full) no increment is possible increment error
            if (dim == contraction_index_num-1){
	      //cuPrintf("NOT DONE -> FALSE\n");
              not_done = false;
              break;
            }

            // make this and all previous dimensions zero
            for (int dim_prev=dim; dim_prev>=0 ; dim_prev--){
              d_contraction_ind[dim_prev] = 0;
            }
            // increment next dimension
            d_contraction_ind[dim+1]++;
            break;
          }
        }
      }
    }while(not_done);
  }
}




__global__ void printData(double* data, size_t count, size_t id){
  cuPrintf("printData id %d", id);
  for(int i=0; i<6; i++){
    double tmp=data[i];
    cuPrintf("data[%d] = %e\n", i, tmp);
  }
}

