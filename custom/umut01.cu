/*
 * author: ck
 * created: 16.02.2012
 * advisor: atc
 */

#include <string.h>
#include <sstream>

#include "../common/utils.cuh"



#include "cutil_inline.h"
#include "../common/kernels.cuh"
#include "../common/cuPrintf.cuh"


void call_calculate_C_mops(size_t ndims, size_t operand_num, operands* ops, std::string output_tensor, bool print, int* d_to_power = NULL){
  calculate_C_mops<<<NUM_BLOCKS, THREADS_FOR_BLOCK>>>((size_t) ndims,
						      (size_t) (operand_num),

						      (size_t**) (ops->d_strides_operand_pointers),

						      (size_t*) (get_d_obj_strides()[output_tensor]),
						      (size_t*) (get_d_obj_cards()["F"]),

						      (size_t**) (ops->d_cards_operand_pointers),
						      (double**) (ops->d_operand_pointers),

						      (double*) (get_d_obj_data()[output_tensor]),
						      //(double*) (get_d_obj_data()["Z0"]),
						      (size_t) (h_objs[output_tensor]->element_number),
						      (size_t) 1,
						      print, 1,
						      d_to_power);
}


void umut01(int nlhs, mxArray *plhs[], int nrhs, const mxArray *prhs[], bool is_parallel){
  // prepare model elements  //////////////////////////////////////////////////////

  size_t op_iter_count = ((double *)mxGetData(prhs[0]))[0];
  mxChar* V_card_sym = mxGetChars(prhs[1]);
  size_t ndims = mxGetNumberOfElements(prhs[1]);
  double* V_cards = (double*) mxGetData(prhs[2]);
  size_t p = ((double *)mxGetData(prhs[3]))[0];


  size_t max_v = mxGetM(prhs[4]);
  size_t max_alpha = mxGetN(prhs[4]);
  bool* R = (bool*) malloc( sizeof(bool) * max_v * max_alpha); // dynamic allocation may not be initialized with = {0} syntax
  if(COUT) std::cout << "init R" << std::endl;
  for (size_t i=0; i<max_v; i++){
    for (size_t j=0; j<max_alpha; j++){
      R[i + j*max_v] = (bool) (((double *)mxGetData(prhs[4]))[i + j*max_v]);
      if(COUT) std::cout << R[i + j*max_v] << " ";
    }
    if(COUT) std::cout << std::endl;
  }

  std::vector<m_tensor> observed_elements;
  //size_t observed_element_num = max_v;
  size_t m_index=5;
  for (size_t t=0; t<max_v; t++){ // need to fill in v number of observed_elements
    const mxArray* m_observed_cards = prhs[m_index];
    m_index++;
    const mxArray* m_observed_data = prhs[m_index];
    m_index++;

    m_tensor tmp_m_tensor;
    tmp_m_tensor.is_updateable = false; // used with latent tensors only

    size_t m_observed_cards_elnum = mxGetNumberOfElements(m_observed_cards);
    tmp_m_tensor.cards_char = (char*) malloc( m_observed_cards_elnum + 1 );
    for (size_t i=0; i<=m_observed_cards_elnum ; i++)
      if ( i == m_observed_cards_elnum )
        tmp_m_tensor.cards_char[i] = '\0';
      else
        tmp_m_tensor.cards_char[i] = (char) mxGetChars(m_observed_cards)[i] ;

    if ( mxGetNumberOfElements(m_observed_data) == 0 ){
      // tensor init data is not given
      tmp_m_tensor.data = NULL;
    }else{
      // tensor init data is given, save pointer
      tmp_m_tensor.data = (double*) mxGetData(m_observed_data);
      if (COUT) std::cout << "found factor with init data. Data size " << mxGetNumberOfElements(m_observed_data) << std::endl;
    }

    observed_elements.push_back(tmp_m_tensor);
  }


  std::vector<m_tensor> latent_elements;
  //size_t latent_element_num = max_v;
  for (size_t t=0; t<max_alpha; t++){ // need to fill in alpha number of latent_elements
    const mxArray* m_latent_cards = prhs[m_index];
    m_index++;
    const mxArray* m_latent_data = prhs[m_index];
    m_index++;

    m_tensor tmp_m_tensor;
    tmp_m_tensor.is_updateable = (bool) (((double *)mxGetData(prhs[m_index]))[0]);
    m_index++;

    size_t m_latent_cards_elnum = mxGetNumberOfElements(m_latent_cards);
    tmp_m_tensor.cards_char = (char*) malloc( m_latent_cards_elnum + 1 );
    for (size_t i=0; i<=m_latent_cards_elnum ; i++)
      if ( i == m_latent_cards_elnum )
        tmp_m_tensor.cards_char[i] = '\0';
      else
        tmp_m_tensor.cards_char[i] = (char) mxGetChars(m_latent_cards)[i] ;

    if ( mxGetNumberOfElements(m_latent_data) == 0 ){
      // tensor init data is not given
      tmp_m_tensor.data = NULL;
    }else{
      // tensor init data is given, save pointer
      tmp_m_tensor.data = (double*) mxGetData(m_latent_data);
      if (COUT) std::cout << "found factor with init data. Data size " << mxGetNumberOfElements(m_latent_data) << std::endl;
    }

    latent_elements.push_back(tmp_m_tensor);
  }


  // prepare cards_numeric indices of model elements
  // input arrives like so:
  // A['i','k'], B['k', 'j'], C['i','j'] where V is ['i','k','j'] = [2 3 4]
  // here we convert indices to internal format:
  // A[2, 3, 0], B[0, 3, 4], C[2, 0, 4]
  for (size_t m=0; m<observed_elements.size(); m++){
    assign_m_tensor_cards_numeric(&(observed_elements[m]), V_card_sym, V_cards, ndims);
  }
  for (size_t m=0; m<latent_elements.size(); m++){
    assign_m_tensor_cards_numeric(&(latent_elements[m]), V_card_sym, V_cards, ndims);
  }


  if (COUT) {
    print_model_elements_text(&observed_elements, "printing observed model elements");
    print_model_elements_text(&latent_elements, "printing latent model elements");
  }

  // now all tensors have correct internal cardinalities.
  // all numeric cardinality arrays (m_tensor.char_numeric) are of same size as V
  // -> ndims





  // prepare output tensor in matlab  //////////////////////////////////////////////////////

  std::vector<double*> output_data_ptr;


  for (size_t t=0; t<latent_elements.size(); t++){
    mwSize argMatDims[ndims];
    for (size_t i=0; i<ndims; i++) {
      size_t val = latent_elements[t].cards_numeric[i];
      if (val == 0) argMatDims[i] = 1; // MATLAB needs to get 1 instead of 0
      else          argMatDims[i] = val;
    }

    plhs[t] = mxCreateNumericArray(ndims, argMatDims, mxDOUBLE_CLASS, mxREAL);

    output_data_ptr.push_back( (double*) mxGetPr(plhs[t]) );
  }




  // prepare host memory for tensors  ///////////////////////////////////////////////////////

  h_full_cardinalities = (size_t*) calloc(ndims, sizeof(size_t)); // defined in mct_tensorop_utils.cuh

  ///// cards_numeric are alligned according to the V cardinalities ///// above //
  for (size_t dim=0; dim<ndims; dim++){ // for each dimension
    size_t max_dim_card = 0;

    for (size_t t=0; t<observed_elements.size(); t++){ // for each model
      for (size_t card=0; card<strlen(observed_elements[t].cards_char); card++){ // for each dimension of the model
        if (observed_elements[t].cards_char[card] == V_card_sym[dim]){ // if this dimension character matches current dimension's
          size_t tensor_dim_card = observed_elements[t].cards_numeric[dim]; //see above//
          if ( max_dim_card < tensor_dim_card )
            max_dim_card = tensor_dim_card;
          break; // only one dimension of each model can match with current dimension
        }
      }
    }

    for (size_t t=0; t<latent_elements.size(); t++){ // for each model
      for (size_t card=0; card<strlen(latent_elements[t].cards_char); card++){ // for each dimension of the model
        if (latent_elements[t].cards_char[card] == V_card_sym[dim]){ // if this dimension character matches current dimension's
          size_t tensor_dim_card = latent_elements[t].cards_numeric[dim]; //see above//
          if ( max_dim_card < tensor_dim_card )
            max_dim_card = tensor_dim_card;
          break; // only one dimension of each model can match with current dimension
        }
      }
    }

    h_full_cardinalities[dim] = max_dim_card;
  }


  if(COUT)
    for (int i=0; i<ndims; i++)
      std::cout << "h_full_cardinalities " << i << " " << h_full_cardinalities[i] << std::endl;


  // initialize random seed for random initialization of objects
  //srand((unsigned)time(NULL));
  srand(123);

  std::vector<ct> X_tensors;
  std::vector<ct> A_tensors;
  std::vector<ct> hat_X_tensors;

  for (size_t el=0; el<observed_elements.size(); el++){
    ct tmp_ct_X;
    ct tmp_ct_A;
    ct tmp_ct_hatX;

    size_t X_card[ndims];
    for (size_t i=0; i<ndims; i++) X_card[i] = observed_elements[el].cards_numeric[i];

    std::stringstream x, hatx, xa;
    x << "Host X" << el;
    hatx << "Host hatX" << el;
    xa << "Host A_X" << el;
    prepareHostTensorFromCpp(&tmp_ct_X, observed_elements[el].data, X_card, ndims, x.str().c_str(), true); // init with given data, if null init with rand
    prepareHostTensorFromCpp(&tmp_ct_A, NULL, X_card, ndims, xa.str().c_str(), false, true); // rand=false, init_to_one=true -> init with 1
    prepareHostTensorFromCpp(&tmp_ct_hatX, NULL, X_card, ndims, hatx.str().c_str(), true);

    X_tensors.push_back(tmp_ct_X);
    A_tensors.push_back(tmp_ct_A);
    hat_X_tensors.push_back(tmp_ct_hatX);
  }


  std::vector<ct> Z_tensors;
  std::vector<ct> Z_update_tensors; // stores each one of sub-update equation results, then all are summed together
  std::vector<ct> D_tensors;
  for (size_t el=0; el<latent_elements.size(); el++){
    ct tmp_ct;

    size_t Z_card[ndims];
    for (size_t i=0; i<ndims; i++) Z_card[i] = latent_elements[el].cards_numeric[i];
    std::stringstream z;
    z << "Host Z" << el;
    prepareHostTensorFromCpp(&tmp_ct, latent_elements[el].data, Z_card, ndims, z.str().c_str(), true); // init with given data, if null init with rand
    Z_tensors.push_back(tmp_ct);

    for (size_t v=0; v<max_v; v++){
      ct tmp_ct_D1;
      ct tmp_ct_D2;

      std::stringstream d1;
      d1 << "Host D1_Z" << el << "X" << v;
      prepareHostTensorFromCpp(&tmp_ct_D1, NULL, Z_card, ndims, d1.str().c_str());

      std::stringstream d2;
      d2 << "Host D2_Z" << el << "X" << v;
      prepareHostTensorFromCpp(&tmp_ct_D2, NULL, Z_card, ndims, d2.str().c_str());

      D_tensors.push_back(tmp_ct_D1);
      D_tensors.push_back(tmp_ct_D2);

      // ct tmp_ct_update;
      // std::stringstream z_update;
      // z_update << "Host Z_update" << el << "X" << v;
      // prepareHostTensorFromCpp(&tmp_ct_update, NULL, Z_card, ndims, z_update.str().c_str(), false, false); // rand=false, init_to_one=false -> init with 0
      // Z_update_tensors.push_back(tmp_ct_update);
    }

    // for summation of division operands
    // ct tmp_ct_D1_sum;
    // ct tmp_ct_D2_sum;
    // std::stringstream d1;
    // d1 << "Host D1_Z" << el << "sum";
    // prepareHostTensorFromCpp(&tmp_ct_D1_sum, NULL, Z_card, ndims, d1.str().c_str());

    // std::stringstream d2;
    // d2 << "Host D2_Z" << el << "sum";
    // prepareHostTensorFromCpp(&tmp_ct_D2_sum, NULL, Z_card, ndims, d2.str().c_str());

    // D_tensors.push_back(tmp_ct_D1_sum);
    // D_tensors.push_back(tmp_ct_D2_sum);
  }


  ct F;
  prepareHostTensorFromCpp(&F, NULL, h_full_cardinalities, ndims, "Host F", true, true, false);


  ///////////////////////////////////////////////////////////////////////////////////////////

  // register & transfer objects to device //////////////////////////////////////////////////

  size_t k=0;
  for (size_t alpha=0; alpha<max_alpha; alpha++){
    for (size_t v=0; v<max_v; v++){
      std::stringstream d_name1;
      d_name1 << "D1_Z" << alpha << "X" << v;
      register_ct( d_name1.str().c_str(), &D_tensors[k]);
      k++;

      std::stringstream d_name2;
      d_name2 << "D2_Z" << alpha << "X" << v;
      register_ct( d_name2.str().c_str(), &D_tensors[k]);
      k++;

      //     std::stringstream name_update;
      //     name_update << "Zup" << alpha << "X" << v;
      //     register_ct( name_update.str().c_str(), &(Z_update_tensors[k]) );
    }

    // std::stringstream d_name1, d_name2;
    // d_name1 << "D1_Z" << alpha << "sum";
    // d_name2 << "D2_Z" << alpha << "sum";
    // register_ct( d_name2.str().c_str(), &D_tensors[k]);
    // k++;
    // register_ct( d_name2.str().c_str(), &D_tensors[k]);
    // k++;
  }

  for (size_t z=0; z<Z_tensors.size(); z++){
    std::stringstream name;
    name << 'Z' << z;
    register_ct( name.str().c_str(), &(Z_tensors[z]) );
  }


  for (size_t x=0; x<X_tensors.size(); x++){
    std::stringstream name;
    name << "X" << x;
    register_ct( name.str().c_str(), &(X_tensors[x]) );

    std::stringstream a_name;
    a_name << "A" << x;
    register_ct( a_name.str().c_str(), &(A_tensors[x]) );

    std::stringstream hat_X_name;
    hat_X_name << "hatX" << x;
    register_ct( hat_X_name.str().c_str(), &(hat_X_tensors[x]) );
  }


  // 'f','i','k','t','m','n'
  ct BC, BZ, FT;
  size_t* BC_card = (size_t*) calloc(ndims, sizeof(size_t));
  // BC(i,k,t) others 0
  BC_card[1] = V_cards[1]; // i
  BC_card[2] = V_cards[2]; // k
  BC_card[3] = V_cards[3]; // t
  prepareHostTensorFromCpp(&BC, NULL, BC_card, ndims, "Host BC");

  size_t* BZ_card = (size_t*) calloc(ndims, sizeof(size_t));
  // BZ(i,k) others 0
  BZ_card[1] = V_cards[1]; // i
  BZ_card[2] = V_cards[2]; // k
  prepareHostTensorFromCpp(&BZ, NULL, BZ_card, ndims, "Host BZ");

  size_t* FT_card = (size_t*) calloc(ndims, sizeof(size_t));
  // FT(i,n) others 0
  FT_card[1] = V_cards[1]; // i
  FT_card[5] = V_cards[5]; // n
  prepareHostTensorFromCpp(&FT, NULL, FT_card, ndims, "Host FT");

  ct X0_ones, X0_tmp1, X0_tmp2;
  size_t X0_cards[ndims];
  for (size_t i=0; i<ndims; i++) X0_cards[i] = observed_elements[0].cards_numeric[i];
  prepareHostTensorFromCpp(&X0_ones, NULL, X0_cards, ndims, "Host X0_ones", false, true);
  prepareHostTensorFromCpp(&X0_tmp1, NULL, X0_cards, ndims, "Host X0_tmp1", false, true);
  prepareHostTensorFromCpp(&X0_tmp2, NULL, X0_cards, ndims, "Host X0_tmp2", false, true);

  ct X1_ones, X1_tmp1, X1_tmp2;
  size_t X1_cards[ndims];
  for (size_t i=0; i<ndims; i++) X1_cards[i] = observed_elements[1].cards_numeric[i];
  prepareHostTensorFromCpp(&X1_ones, NULL, X1_cards, ndims, "Host X1_ones", false, true);
  prepareHostTensorFromCpp(&X1_tmp1, NULL, X1_cards, ndims, "Host X1_tmp1", false, true);
  prepareHostTensorFromCpp(&X1_tmp2, NULL, X1_cards, ndims, "Host X1_tmp2", false, true);


  ct X2_ones, X2_tmp1, X2_tmp2;
  size_t X2_cards[ndims];
  for (size_t i=0; i<ndims; i++) X2_cards[i] = observed_elements[2].cards_numeric[i];
  prepareHostTensorFromCpp(&X2_ones, NULL, X2_cards, ndims, "Host X2_ones", false, true);
  prepareHostTensorFromCpp(&X2_tmp1, NULL, X2_cards, ndims, "Host X2_tmp1", false, true);
  prepareHostTensorFromCpp(&X2_tmp2, NULL, X2_cards, ndims, "Host X2_tmp2", false, true);

  ct ikt;
  size_t* ikt_card = (size_t*) calloc(3, sizeof(size_t));
  prepareHostTensorFromCpp(&ikt, NULL, ikt_card, 3, "Host ikt", false, false, false);

  REGISTER_CT(F);

  REGISTER_CT(BC); REGISTER_CT(BZ); REGISTER_CT(FT);
  REGISTER_CT(X0_ones); REGISTER_CT(X1_ones); REGISTER_CT(X2_ones);
  REGISTER_CT(X0_tmp1); REGISTER_CT(X1_tmp1); REGISTER_CT(X2_tmp1);
  REGISTER_CT(X0_tmp2); REGISTER_CT(X1_tmp2); REGISTER_CT(X2_tmp2);

  REGISTER_CT(ikt);

  if (CUPRINTF == true)
    cudaPrintfInit();

  std::cout << " selam 1 " << std::endl;
  size_t cur_mem;
  if (is_parallel)
    cur_mem = transferToDevice(ndims);

  if( COUT ) std::cout << "transferToDevice " << cur_mem << " bytes " << std::endl;
  ///////////////////////////////////////////////////////////////////////////////////////////

  // perform GCTF operation //////////////////////////////////////////////////////////////////


  std::vector<std::string> sops_1;
  operands ops_1;
  sops_1.push_back("BZ");
  sops_1.push_back("Z3");
  cur_mem = gen_operation_arguments( sops_1, &ops_1, cur_mem );

  std::vector<std::string> sops_2;
  operands ops_2;
  sops_2.push_back("Z0");
  sops_2.push_back("BC");
  cur_mem = gen_operation_arguments( sops_2, &ops_2, cur_mem );

  std::vector<std::string> sops_3;
  operands ops_3;
  sops_3.push_back("X0_tmp1");
  sops_3.push_back("BC");
  cur_mem = gen_operation_arguments( sops_3, &ops_3, cur_mem );

  std::vector<std::string> sops_4;
  operands ops_4;
  sops_4.push_back("X0_tmp2");
  sops_4.push_back("BC");
  cur_mem = gen_operation_arguments( sops_4, &ops_4, cur_mem );

  std::vector<std::string> sops_5;
  operands ops_5;
  sops_5.push_back("Z0");
  sops_5.push_back("FT");
  cur_mem = gen_operation_arguments( sops_5, &ops_5, cur_mem );

  std::vector<std::string> sops_6;
  operands ops_6;
  sops_6.push_back("X2_tmp0");
  sops_6.push_back("FT");
  cur_mem = gen_operation_arguments( sops_6, &ops_6, cur_mem );

  std::vector<std::string> sops_7;
  operands ops_7;
  sops_7.push_back("hatX2");
  sops_7.push_back("FT");
  int to_power_7[2];
  to_power_7[0] = 1-p;
  to_power_7[1] = 1;
  cur_mem = gen_operation_arguments( sops_7, &ops_7, cur_mem, to_power_7 );









  for (int iter=0; iter<op_iter_count; iter++){
    std::cout << "iter " << iter << std::endl;



    // D -> Z0
    // B -> Z1
    // Z -> Z2
    // C -> Z3
    // G -> Z4
    // Y -> Z5
    // F -> Z6
    // T -> Z7








    // update D
    
    // compute x1hat

    // B.*Z -> BZ
    hadamard_mul<<<NUM_BLOCKS, THREADS_FOR_BLOCK>>>( get_d_obj_data()["Z1"],
						     get_d_obj_data()["Z2"],
						     get_d_obj_data()["BZ"],
						     h_objs["BZ"]->element_number,
						     CUPRINTF);

    // BZ(i,k)*C(k,t) -> BC(i,k,t)
    calculate_C_mops<<<NUM_BLOCKS, THREADS_FOR_BLOCK>>>((size_t) 3,
							(size_t) (sops_1.size()),

							(size_t**) (ops_1.d_strides_operand_pointers),

							(size_t*) (get_d_obj_strides()["BC"]),
							(size_t*) (get_d_obj_cards()["ikt"]),

							(size_t**) (ops_1.d_cards_operand_pointers),
							(double**) (ops_1.d_operand_pointers),

							(double*) (get_d_obj_data()["BC"]),
							//(double*) (get_d_obj_data()["Z0"]),
							(size_t) (h_objs["BC"]->element_number),
							(size_t) 1,
							CUPRINTF,1);

    break;

    // X1hat = D*BC;
    calculate_C_mops<<<NUM_BLOCKS, THREADS_FOR_BLOCK>>>((size_t) ndims,
							(size_t) (sops_2.size()),

							(size_t**) (ops_2.d_strides_operand_pointers),

							(size_t*) (get_d_obj_strides()["hatX0"]),
							(size_t*) (get_d_obj_cards()["F"]),

							(size_t**) (ops_2.d_cards_operand_pointers),
							(double**) (ops_2.d_operand_pointers),

							(double*) (get_d_obj_data()["hatX0"]),
							//(double*) (get_d_obj_data()["Z0"]),
							(size_t) (h_objs["hatX0"]->element_number),
							(size_t) 1,
							CUPRINTF,1);


    //arg_D_n_1 =  M1.* X1 .* (X1hat.^(-p));
    hadamard_mul<<<NUM_BLOCKS, THREADS_FOR_BLOCK>>>( get_d_obj_data()["X0"],
						     get_d_obj_data()["hatX0"],
						     get_d_obj_data()["X0_tmp1"],
						     h_objs["X0"]->element_number,
						     CUPRINTF, 1, -p);

    //arg_D_d_1 =  M1.* (X1hat.^(1-p));
    hadamard_mul<<<NUM_BLOCKS, THREADS_FOR_BLOCK>>>( get_d_obj_data()["X0_ones"],
						     get_d_obj_data()["hatX0"],
						     get_d_obj_data()["X0_tmp2"],
						     h_objs["X0"]->element_number,
						     CUPRINTF, 1, 1-p);


    // deltaD_n_1 = arg_D_n_1 * (BC)';
    calculate_C_mops<<<NUM_BLOCKS, THREADS_FOR_BLOCK>>>((size_t) ndims,
							(size_t) (sops_3.size()),

							(size_t**) (ops_3.d_strides_operand_pointers),

							(size_t*) (get_d_obj_strides()["D1_Z0X0"]),
							(size_t*) (get_d_obj_cards()["F"]),

							(size_t**) (ops_3.d_cards_operand_pointers),
							(double**) (ops_3.d_operand_pointers),

							(double*) (get_d_obj_data()["D1_Z0X0"]),
							//(double*) (get_d_obj_data()["Z0"]),
							(size_t) (h_objs["D1_Z0X0"]->element_number),
							(size_t) 1,
							CUPRINTF,1);

    //deltaD_d_1 = arg_D_d_1 * (BC)';
    calculate_C_mops<<<NUM_BLOCKS, THREADS_FOR_BLOCK>>>((size_t) ndims,
							(size_t) (sops_4.size()),

							(size_t**) (ops_4.d_strides_operand_pointers),

							(size_t*) (get_d_obj_strides()["D2_Z0X0"]),
							(size_t*) (get_d_obj_cards()["F"]),

							(size_t**) (ops_4.d_cards_operand_pointers),
							(double**) (ops_4.d_operand_pointers),

							(double*) (get_d_obj_data()["D2_Z0X0"]),
							//(double*) (get_d_obj_data()["Z0"]),
							(size_t) (h_objs["D2_Z0X0"]->element_number),
							(size_t) 1,
							CUPRINTF,1);

    //Compute X3hat
    // FT = F.*T;
    hadamard_mul<<<NUM_BLOCKS, THREADS_FOR_BLOCK>>>( get_d_obj_data()["Z6"],
						     get_d_obj_data()["Z7"],
						     get_d_obj_data()["FT"],
						     h_objs["Z6"]->element_number,
						     CUPRINTF);
    
    // X3hat = D*FT;
    calculate_C_mops<<<NUM_BLOCKS, THREADS_FOR_BLOCK>>>((size_t) ndims,
							(size_t) (sops_5.size()),

							(size_t**) (ops_5.d_strides_operand_pointers),

							(size_t*) (get_d_obj_strides()["hatX2"]),
							(size_t*) (get_d_obj_cards()["F"]),

							(size_t**) (ops_5.d_cards_operand_pointers),
							(double**) (ops_5.d_operand_pointers),

							(double*) (get_d_obj_data()["hatX2"]),
							//(double*) (get_d_obj_data()["Z0"]),
							(size_t) (h_objs["hatX2"]->element_number),
							(size_t) 1,
							CUPRINTF,1);
    
    // arg_D_n_2 =  X3 .* (X3hat.^(-p));
    hadamard_mul<<<NUM_BLOCKS, THREADS_FOR_BLOCK>>>( get_d_obj_data()["X2"],
						     get_d_obj_data()["hatX2"],
						     get_d_obj_data()["X2_tmp0"],
						     h_objs["X2"]->element_number,
						     CUPRINTF, 1, -p);

    // arg_D_d_2 =  X3hat.^(1-p);
    // skip
    // hadamard_mul<<<NUM_BLOCKS, THREADS_FOR_BLOCK>>>( get_d_obj_data()["X2"],
    // 						     get_d_obj_data()["hatX2"],
    // 						     get_d_obj_data()["D2_Z0X2"],
    // 						     h_objs["X2"]->element_number,
    // 						     CUPRINTF, 1, 1-p);


    //deltaD_n_2 = arg_D_n_2 * (FT)';
    call_calculate_C_mops(ndims, 2, &ops_6, "D1_Z0X2", CUPRINTF);


    //deltaD_d_2 = arg_D_d_2 * (FT)';
    call_calculate_C_mops(ndims, 2, &ops_7, "D2_Z0X2", CUPRINTF);

    //D = D.* ( (deltaD_n_1 + deltaD_n_2 ) ./ (deltaD_d_1 + deltaD_d_2 ));
    hadamard_sum<<<NUM_BLOCKS, THREADS_FOR_BLOCK>>>( get_d_obj_data()["D1_Z0X0"],
						     get_d_obj_data()["D1_Z0X2"],
						     get_d_obj_data()["D1_Z0X0"],
						     h_objs["D1_Z0X0"]->element_number,
						     CUPRINTF);
    hadamard_sum<<<NUM_BLOCKS, THREADS_FOR_BLOCK>>>( get_d_obj_data()["D2_Z0X0"],
						     get_d_obj_data()["D2_Z0X2"],
						     get_d_obj_data()["D2_Z0X0"],
						     h_objs["D2_Z0X0"]->element_number,
						     CUPRINTF);
    hadamard_div<<<NUM_BLOCKS, THREADS_FOR_BLOCK>>>( get_d_obj_data()["D1_Z0X0"],
						     get_d_obj_data()["D2_Z0X0"],
						     get_d_obj_data()["D1_Z0X0"],
						     h_objs["D1_Z0X0"]->element_number,
						     CUPRINTF);
    hadamard_mul<<<NUM_BLOCKS, THREADS_FOR_BLOCK>>>( get_d_obj_data()["Z0"],
						     get_d_obj_data()["D1_Z0X0"],
						     get_d_obj_data()["Z0"],
						     h_objs["Z0"]->element_number,
						     CUPRINTF);



    break;

    // // calculate all hatX_v and A_v
    // for (size_t alpha=0; alpha<max_alpha; alpha++){
    //   if ( latent_elements[alpha].is_updateable == false) continue;



    //   // update all hatX
    //   for ( size_t cur_v=0; cur_v<max_v; cur_v++){
    //     std::stringstream hat_Xv;
    //     hat_Xv << "hatX" << cur_v;


    //     operands ops_Z0_ZN_Xhat;
    //     std::vector<std::string> z_tensors_str;
    //     for (size_t tmp_alpha=0; tmp_alpha<max_alpha; tmp_alpha++){
    //       if ( R[cur_v + tmp_alpha*max_v] == false )  continue;

    //       std::stringstream name;
    //       name << 'Z' << tmp_alpha;
    //       z_tensors_str.push_back(name.str());
    //     }

    // 	std::cout << "operand num z_tensors_str.size() " << z_tensors_str.size() << std::endl;
    // 	for( size_t i=0; i<z_tensors_str.size(); i++){
    // 	  std::cout << "z_tensors_str[" << i << "] = " << z_tensors_str[i] << std::endl;
    // 	}
	
    //     cur_mem = gen_operation_arguments(z_tensors_str, &ops_Z0_ZN_Xhat, cur_mem);

    //     // Z0 * Z1 * ... * ZN -> Xhat
    //     //std::cout << "Z0 * Z1 * ... * ZN -> " << hat_Xv.str() << std::endl;
    //     calculate_C_mops<<<NUM_BLOCKS, THREADS_FOR_BLOCK>>>((size_t) ndims,
    //                                                         (size_t) (z_tensors_str.size()),

    //                                                         (size_t**) (ops_Z0_ZN_Xhat.d_strides_operand_pointers),

    //                                                         (size_t*) (get_d_obj_strides()[hat_Xv.str()]),
    //                                                         (size_t*) (get_d_obj_cards()["F"]),

    //                                                         (size_t**) (ops_Z0_ZN_Xhat.d_cards_operand_pointers),
    //                                                         (double**) (ops_Z0_ZN_Xhat.d_operand_pointers),

    //                                                         (double*) (get_d_obj_data()[hat_Xv.str()]),
    //                                                         //(double*) (get_d_obj_data()["Z0"]),
    //                                                         (size_t) (h_objs[hat_Xv.str()]->element_number),
    //                                                         (size_t) 1,
    //                                                         CUPRINTF,1);

    // 	////std::cout << " z0 * z1 * .. * zn -> " << hat_Xv << " done " << std::endl;
    //     std::stringstream Xv;
    //     Xv << 'X' << cur_v ;

    //     std::stringstream Av;
    //     Av << 'A' << cur_v;

    //     hadamard_div<<<NUM_BLOCKS, THREADS_FOR_BLOCK>>>( get_d_obj_data()[hat_Xv.str().c_str()],
    //                                                      get_d_obj_data()[Xv.str().c_str()],
    //                                                      get_d_obj_data()[Av.str().c_str()],
    //                                                      h_objs[Av.str().c_str()]->element_number,
    //                                                      CUPRINTF,
    //                                                      p, 1);
    //   }
    //   return;








    //   // for each Xv
    //   for (size_t cur_v=0; cur_v<max_v; cur_v++){
    //     if ( R[cur_v + alpha*max_v] == false ) continue; // if this Xv does not have this Zalpha dothing to do

    //     std::stringstream hat_Xv;
    //     hat_Xv << "hatX" << cur_v;

    //     // calculate D1_Zalpha_Xv
    //     std::stringstream d1;
    //     d1 << "D1_Z" << alpha << "X" << cur_v;

    //     // calculate D2_Zalpha_Xv
    //     std::stringstream d2;
    //     d2 << "D2_Z" << alpha << "X" << cur_v;

    //     std::stringstream Av;
    //     Av << 'A' << cur_v;


    //     operands ops_A;
    //     operands ops_M;


    //     std::vector<std::string> tmp_A;
    //     std::vector<std::string> tmp_M;

    //     tmp_A.push_back(Av.str());
    //     tmp_M.push_back(hat_Xv.str());


    //     for (size_t other_z=0; other_z < max_alpha; other_z++){
    //       //std::cout << " process alpha " << alpha << " other_z " << other_z << std::endl;
    //       if (other_z == alpha || R[cur_v + other_z*max_v] == false ) continue;

    //       std::stringstream other_z_name;
    //       other_z_name << "Z" << other_z;

    //       tmp_A.push_back(other_z_name.str());
    //       tmp_M.push_back(other_z_name.str());
    //       //std::cout << "pushing to tmp_A and tmp_M: " << other_z_name.str()  << std::endl;

    //     }

    // 	//std::cout << "operand num tmp_A.size() " << tmp_A.size() << std::endl;
    // 	for( size_t i=0; i<tmp_A.size(); i++){
    // 	}


    //     cur_mem = gen_operation_arguments(tmp_A, &ops_A, cur_mem);

    //     //oc_push_back(&operation_chain, GMULT, ndims, Av.str().c_str(), other_z_name.str().c_str(), d1.str().c_str(), is_parallel);
    //     calculate_C_mops<<<NUM_BLOCKS, THREADS_FOR_BLOCK>>>((size_t) ndims,
    //                                                         (size_t) (tmp_A.size()),

    //                                                         (size_t**) (ops_A.d_strides_operand_pointers),

    //                                                         (size_t*) (get_d_obj_strides()[d1.str().c_str()]),
    //                                                         (size_t*) (get_d_obj_cards()["F"]),

    //                                                         (size_t**) (ops_A.d_cards_operand_pointers),
    //                                                         (double**) (ops_A.d_operand_pointers),

    //                                                         (double*) (get_d_obj_data()[d1.str().c_str()]),
    //                                                         //(double*) (get_d_obj_data()["Z0"]),
    //                                                         (size_t) (h_objs[d1.str()]->element_number),
    //                                                         (size_t) 1,
    //                                                         CUPRINTF,2);

    //     //oc_push_back(&operation_chain, GMULT, ndims, hat_Xv.str().c_str(), other_z_name.str().c_str(), d2.str().c_str(), is_parallel, "F", p+1, 1);

    // 	int to_power[tmp_M.size()];
    // 	to_power[0]=p+1;
    // 	for (size_t i=0; i<tmp_M.size(); i++){
    // 	  to_power[i] = 0;
    // 	}
    //     cur_mem = gen_operation_arguments(tmp_M, &ops_M, cur_mem, to_power);

    //     calculate_C_mops<<<NUM_BLOCKS, THREADS_FOR_BLOCK>>>((size_t) ndims,
    //                                                         (size_t) (tmp_M.size()),

    //                                                         (size_t**) (ops_M.d_strides_operand_pointers),

    //                                                         (size_t*) (get_d_obj_strides()[d2.str().c_str()]),
    //                                                         (size_t*) (get_d_obj_cards()["F"]),

    //                                                         (size_t**) (ops_M.d_cards_operand_pointers),
    //                                                         (double**) (ops_M.d_operand_pointers),

    //                                                         (double*) (get_d_obj_data()[d2.str().c_str()]),
    //                                                         //(double*) (get_d_obj_data()["Z0"]),
    //                                                         (size_t) (h_objs[d2.str()]->element_number),
    //                                                         (size_t) 1,
    //                                                         CUPRINTF,3,
    // 							    ops_M.d_to_power
    // 							    );


    //   }







    //   // sum D1_Zalpha_Xv and D2_Zalpha_Xv for all v to update Zalpha
    //   std::stringstream D1_Zalpha_sum, D2_Zalpha_sum; // will sum into these

    //   bool first = true;
    //   for (size_t v=0; v<max_v; v++){
    //     if ( R[v + alpha*max_v] ){
    //       if ( first ){
    //         D1_Zalpha_sum << "D1_Z" << alpha << "X" << v;
    //         D2_Zalpha_sum << "D2_Z" << alpha << "X" << v;
    //         first = false;
    //       }else{
    //         std::stringstream other_d1, other_d2;
    //         other_d1 << "D1_Z" << alpha << "X" << v;
    //         other_d2 << "D2_Z" << alpha << "X" << v;

    // 	    hadamard_sum<<<NUM_BLOCKS, THREADS_FOR_BLOCK>>>(get_d_obj_data()[D1_Zalpha_sum.str().c_str()],
    // 							    get_d_obj_data()[other_d1.str().c_str()],
    // 							    get_d_obj_data()[D1_Zalpha_sum.str().c_str()],
    // 							    h_objs[D1_Zalpha_sum.str().c_str()]->element_number,
    // 							    CUPRINTF);

    // 	    hadamard_sum<<<NUM_BLOCKS, THREADS_FOR_BLOCK>>>(get_d_obj_data()[D2_Zalpha_sum.str().c_str()],
    // 							    get_d_obj_data()[other_d2.str().c_str()],
    // 							    get_d_obj_data()[D2_Zalpha_sum.str().c_str()],
    // 							    h_objs[D2_Zalpha_sum.str().c_str()]->element_number,
    // 							    CUPRINTF);

    //       }
    //     }
    //   }

    //   hadamard_div<<<NUM_BLOCKS, THREADS_FOR_BLOCK>>>( get_d_obj_data()[D1_Zalpha_sum.str().c_str()],
    // 						       get_d_obj_data()[D2_Zalpha_sum.str().c_str()],
    // 						       get_d_obj_data()[D1_Zalpha_sum.str().c_str()],
    // 						       h_objs[D1_Zalpha_sum.str().c_str()]->element_number,
    // 						       CUPRINTF);

    //   std::stringstream Zalpha;
    //   Zalpha << 'Z' << alpha ;
    //   hadamard_mul<<<NUM_BLOCKS, THREADS_FOR_BLOCK>>>( get_d_obj_data()[Zalpha.str().c_str()],
    // 						       get_d_obj_data()[D1_Zalpha_sum.str().c_str()],
    // 						       get_d_obj_data()[Zalpha.str().c_str()],
    // 						       h_objs[Zalpha.str().c_str()]->element_number,
    // 						       CUPRINTF);

    //}

  }





  // check if kernel execution generated and error
  cutilCheckMsg("Kernel execution failed");
  hipDeviceSynchronize();

  if ( CUPRINTF == true ){
    cudaPrintfDisplay(stdout, true);
    cudaPrintfEnd();
  }









  ///////////////////////////////////////////////////////////////////////////////////////////

  // transfer results to matlab /////////////////////////////////////////////////////////////

  if ( is_parallel ){
    for (size_t z=0; z<latent_elements.size(); z++){
      std::stringstream Zn;
      Zn << 'Z' << z;
      transferFromDevice(output_data_ptr[z], Zn.str().c_str());
    }
    //transferFromDevice(m_Z1, "Z1");
    //transferFromDevice(m_Z2, "Z2");
  }else{
    for (size_t z=0; z<latent_elements.size(); z++){
      memcpy(output_data_ptr[z], Z_tensors[z].data, Z_tensors[z].mem_size);
    }
    //memcpy(m_Z1, Z1.data, Z1.mem_size);
    //memcpy(m_Z2, Z2.data, Z2.mem_size);
  }

  ///////////////////////////////////////////////////////////////////////////////////////////


  // reset device
  if ( is_parallel )
    resetDevice();

  hipDeviceReset();
}
