#include "hip/hip_runtime.h"
/*
 * author: ck
 * created: 05.08.2011
 * advisor: atc
 */

//#include "mex.h"
#include "hipblas.h"

#include <iostream>
#include <algorithm>
//#include <vector>

#include <string.h>

#include "mct_tensorop_utils.cuh"

#include "mct_tensorop_gpu.cuh"
#include "mct_tensorop_cpp.cuh"

#include <time.h>

#define REGISTER_CT(obj) register_ct(#obj, &obj)

enum op_type {
  tensor_gpu,
  tensor_cpp,
  nmf_gpu,
  nmf_cpp,
  num_of_op_types
};


void tensorop(int nlhs, mxArray *plhs[], int nrhs, const mxArray *prhs[], op_type opt){
  if( nlhs != 1 ){
    std::cout << "mct: tensor operation requires exactly one output argument" << std::endl;
    // print help;
    return;
  }

  if ( nrhs != (6+1) ){
    std::cout << "mct: tensor operation requires 6 arguments. "
              << "A, dimensions of A, B, dimensions of B, dimensions of C,"
              << " use_multiplication(1 uses multiplication, 0 uses division)"
              << std::endl;
    return;
  }

  const mxArray* m_A_data = prhs[1];
  const mxArray* m_A_card = prhs[2];

  const mxArray* m_B_data = prhs[3];
  const mxArray* m_B_card = prhs[4];

  const mxArray* m_C_card = prhs[5];

  size_t use_multiplication = ((double *)mxGetData(prhs[6]))[0];

  // assume same size cardinalities for all objects
  size_t ndims = mxGetNumberOfElements(m_A_card);

  // if all cardinalities of A,B,C are the same, we have hadamard operation
  bool isHadamard=true;

  // full_cardinalities define maximum possible cardinalities for all dimensions
  h_full_cardinalities = (size_t*) calloc(ndims, sizeof(size_t)); // defined in mct_tensorop_utils.cuh
  for( size_t i=0; i<ndims; i++){
    double m_A_card_i = ((double *)mxGetData(m_A_card))[i];
    double m_B_card_i = ((double *)mxGetData(m_B_card))[i];
    double m_C_card_i = ((double *)mxGetData(m_C_card))[i];

    h_full_cardinalities[i] = std::max(std::max(m_A_card_i,m_B_card_i), m_C_card_i);

    if ( m_A_card_i != m_B_card_i || m_B_card_i != m_C_card_i ) isHadamard=false;
  }

  if (COUT) if (isHadamard) std::cout << "HADAMARD OPERATION" << std::endl;


  // prepare output tensor in matlab  //////////////////////////////////////////////////////

  size_t non_zero_dim_number=0;
  for (size_t i=0; i<ndims; i++){
    //std::cout << " non_zero_dim_number loop " << i ;
    double tmpdimcard = ((double*)mxGetData(m_C_card))[i];
    if(tmpdimcard != 0) {
      non_zero_dim_number++;
      //std::cout  << " tmpdimcard " << tmpdimcard << std::endl;
      //m_C_mem_size *= tmpdimcard;
    }
  }

  mwSize argMatDims[non_zero_dim_number];
  size_t argMatDims_ind=0;
  size_t C_elnum=1;
  //std::cout << "C tensor init argMatDims with size " << non_zero_dim_number << std::endl;
  //<< " m_C_mem_size " << m_C_mem_size << std::endl;


  for (size_t i=0; i<ndims; i++){
    double val=((double*)mxGetData(m_C_card))[i];
    //std::cout << "C tensor argMatDims[" << i << "] = " << val << " ";
    if ( val != 0){ // skip dimensions with 0 cardinality
      //std::cout << " assign " << std::endl;
      argMatDims[argMatDims_ind] = val;
      argMatDims_ind++;
      C_elnum *= val;
    }else{
      //std::cout << " not assign " << std::endl;
    }
  }


  /*
    for (size_t i=0; i<ndims; i++){
    double val=((double*)mxGetData(m_C_card))[i];
    //std::cout << "C tensor argMatDims[" << i << "] = " << val << " ";
    //std::cout << " assign " << std::endl;
    if ( val == 0){
    argMatDims[argMatDims_ind] = 1;
    }else{
    argMatDims[argMatDims_ind] = val;
    C_elnum *= val;
    }
    argMatDims_ind++;
    }
  */

  if (non_zero_dim_number==0){
    // contraction on all dimensions
    // will result in a single number
    mwSize argMatDim[1] = {1};
    plhs[0] = mxCreateNumericArray(1,argMatDim,mxDOUBLE_CLASS,mxREAL);
  }else{
    plhs[0] = mxCreateNumericArray(non_zero_dim_number,argMatDims,mxDOUBLE_CLASS,mxREAL);
  }

  // mwSize alldims[ndims];
  // for (size_t i=0; i<ndims; i++){
  //   alldims[i]=((double*)mxGetData(m_C_card))[i];
  // }

  // plhs[0] = mxCreateNumericArray(ndims,alldims,mxDOUBLE_CLASS,mxREAL);

  double* m_C = (double*) mxGetPr(plhs[0]);

  ///////////////////////////////////////////////////////////////////////////////////////////

  // prepare host memory for tensors  ///////////////////////////////////////////////////////

  ct h_A, h_B, h_C, h_F;
  prepareHostTensor(&h_A, m_A_data, m_A_card, (const char*) "Host A");
  prepareHostTensor(&h_B, m_B_data, m_B_card, (const char*)"Host B");
  // NULL initiates data with zero
  prepareHostTensorFromCpp(&h_F, NULL, h_full_cardinalities, ndims, "Host F");

  // read C cardinalities from matlab side
  size_t* tmp_arr = (size_t*) malloc(sizeof(size_t)*ndims);
  for ( size_t i=0; i<ndims; i++) tmp_arr[i] = (size_t) (((double*) mxGetData(m_C_card))[i]);

  prepareHostTensorFromCpp(&h_C, NULL, tmp_arr, ndims, "Host C");



  // prepare range permutation vector //////////////////////////////////////////////////////
  size_t h_zero_cardinality_dim_tuple_size_C = 0;
  size_t h_zero_cardinality_dim_tuples_C_element_number = 0;
  size_t* h_zero_cardinality_dim_tuples_C = NULL;

  if ( isHadamard == false){
    std::vector<size_t> zero_cardinality_dims;
    //std::vector<size_t> non_zero_cardinality_dims;
    for ( size_t dim=0; dim<ndims; dim++ ){
      if ( h_C.cardinalities[dim] == 0 && h_F.cardinalities[dim] != 0 ){
        zero_cardinality_dims.push_back(h_F.cardinalities[dim]);
      }
      // else{
      //   non_zero_cardinality_dims.push_back(h_F.cardinalities[dim]);
      // }
    }

    // std::cout << "non_zero_cardinality_dims" << std::endl;
    // for ( size_t j=0; j<non_zero_cardinality_dims.size(); j++){
    //   std::cout << non_zero_cardinality_dims.at(j) << std::endl;
    // }

    if ( COUT ) {
      std::cout << "zero_cardinality_dims" << std::endl;
      for ( size_t j=0; j<zero_cardinality_dims.size(); j++){
        std::cout << zero_cardinality_dims.at(j) << std::endl;
      }
    }

    h_zero_cardinality_dim_tuple_size_C = zero_cardinality_dims.size();


    //h_zero_cardinality_dim_tuples_C_element_number; // set by gen_range_permutation

    h_zero_cardinality_dim_tuples_C =
      gen_range_permutation(zero_cardinality_dims,
                            &(h_zero_cardinality_dim_tuples_C_element_number));
  }



  if ( opt == tensor_gpu ) {
    mct_tensorop_gpu(isHadamard, h_A, h_B, h_C, m_C, h_F, ndims, h_zero_cardinality_dim_tuples_C_element_number, h_zero_cardinality_dim_tuples_C, h_zero_cardinality_dim_tuple_size_C, use_multiplication);
  }else{  // operate on CPU
    mct_tensorop_cpp(isHadamard, h_A, h_B, h_C, m_C, h_F, ndims, h_zero_cardinality_dim_tuples_C_element_number, h_zero_cardinality_dim_tuples_C);
  }

}
















void oc_push_back(std::vector<operation>* operation_chain, bool isHadamard, bool use_multiplication, size_t ndims, std::string A, std::string B, std::string C, op_type opt, std::string F="F"){
  operation oc;
  oc.isHadamard = isHadamard;
  oc.use_multiplication = use_multiplication;
  oc.ndims = ndims;
  oc.A = A;
  oc.B = B;
  oc.C = C;
  oc.F = F;
  oc.result_in_F = false;

  if (opt == nmf_gpu){
    oc.operate = &mct_tensorop_gpu_keys;
  }else if (opt == nmf_cpp){
    oc.operate = &mct_tensorop_cpp_keys;
  }


  operation_chain->push_back(oc);
}



void nmfop(int nlhs, mxArray *plhs[], int nrhs, const mxArray *prhs[], op_type opt){
  if( nlhs != 2 ){
    std::cout << "mct: NMF operation requires exactly two output arguments" << std::endl;
    // print help;
    return;
  }

  if ( nrhs != (2+1) ){
    std::cout << "mct: NMF operation requires 2 arguments. "
              << "X, M"
              << std::endl;
    return;
  }

  const mxArray* m_X_data = prhs[1];
  const mxArray* m_M_data = prhs[2];

  if (mxGetNumberOfDimensions(m_X_data) != 2 ){
    std::cout << "mct: NMF requires two dimensional data (matrix) input" << std::endl;
    std::cout << "mct: found data dimenstions: " << mxGetNumberOfDimensions(m_X_data) << std::endl;
    // print help;
    return;
  }

  // prepare output tensor in matlab  //////////////////////////////////////////////////////

  size_t card_dim0 = (mxGetDimensions(m_X_data))[0];
  size_t card_dim1 = (mxGetDimensions(m_X_data))[1];
  mwSize argMatDims_z1[2] = { card_dim0, card_dim1 };
  mwSize argMatDims_z2[2] = { card_dim1, card_dim1 };

  plhs[0] = mxCreateNumericArray(2,argMatDims_z1,mxDOUBLE_CLASS,mxREAL);
  plhs[1] = mxCreateNumericArray(2,argMatDims_z2,mxDOUBLE_CLASS,mxREAL);

  double* m_Z1 = (double*) mxGetPr(plhs[0]);
  double* m_Z2 = (double*) mxGetPr(plhs[1]);

  // prepare host memory for tensors  ///////////////////////////////////////////////////////
  size_t ndims = 3;

  // full_cardinalities define maximum possible cardinalities for all dimensions
  //size_t full_cardinalities[3] = { card, card, card};

  h_full_cardinalities = (size_t*) calloc(ndims, sizeof(size_t)); // defined in mct_tensorop_utils.cuh
  h_full_cardinalities[0] = card_dim0;
  h_full_cardinalities[1] = card_dim1;
  h_full_cardinalities[2] = card_dim1;

  if(COUT)
    for (int i=0; i<3; i++)
      std::cout << "h_full_cardinalities " << i << " " << h_full_cardinalities[i] << std::endl;


  // initialize random seed for random initialization of objects
  //srand((unsigned)time(NULL));
  srand(123);
  ct Z1, Z2, Xhat, X, D1_z1, D1_z2, D2_z1, D2_z2, A, M, F;

  mwSize m_X_card_size = 3;

  mxArray* m_X_card = mxCreateNumericArray(1,&m_X_card_size,mxDOUBLE_CLASS,mxREAL);

  size_t X_card[3] = {card_dim0, 0, card_dim1};
  for (size_t i=0; i<3; i++)
    mxGetPr(m_X_card)[i] = X_card[i];


  prepareHostTensor(&X, m_X_data, m_X_card, (const char*) "Host X");
  prepareHostTensor(&M, m_M_data, m_X_card, (const char*) "Host M");
  prepareHostTensorFromCpp(&A, NULL, X_card, ndims, (const char*) "Host A"); // init with 0
  prepareHostTensorFromCpp(&Xhat, NULL, X_card, ndims, (const char*) "Host Xhat");
  prepareHostTensorFromCpp(&F, NULL, h_full_cardinalities, ndims, "Host F");

  size_t Z1_card[3] = {card_dim0 , card_dim1, 0};
  prepareHostTensorFromCpp(&Z1, NULL, Z1_card, ndims, (const char*) "Host Z1", true);
  size_t Z2_card[3] = {0 , card_dim1, card_dim1};
  prepareHostTensorFromCpp(&Z2, NULL, Z2_card, ndims, (const char*) "Host Z2", true);

  // used first as C must be zeroed for cpp to work
  prepareHostTensorFromCpp(&D1_z1, NULL, Z1_card, ndims, (const char*) "Host D1_z1"); 
  prepareHostTensorFromCpp(&D1_z2, NULL, Z2_card, ndims, (const char*) "Host D1_z2");

  prepareHostTensorFromCpp(&D2_z1, NULL, Z1_card, ndims, (const char*) "Host D2_z1");
  prepareHostTensorFromCpp(&D2_z2, NULL, Z2_card, ndims, (const char*) "Host D2_z2");

  if(PRINT_CT){
    print_ct("random Z1 init", &Z1, true);
    print_ct("random Z2 init", &Z2, true);
    print_ct("target X (cpp side)", &X, true);
    print_ct("F (cpp side)", &F, true);
  }

  ///////////////////////////////////////////////////////////////////////////////////////////


  // register & transfer objects to device //////////////////////////////////////////////////

  REGISTER_CT(Z1); REGISTER_CT(Z2); REGISTER_CT(Xhat); REGISTER_CT(X); REGISTER_CT(D1_z1); REGISTER_CT(D1_z2); REGISTER_CT(D2_z1); REGISTER_CT(D2_z2); REGISTER_CT(A); REGISTER_CT(M); REGISTER_CT(F);

  if (opt==nmf_gpu)
    transferToDevice(ndims);

  ///////////////////////////////////////////////////////////////////////////////////////////


  // perform NMF operation //////////////////////////////////////////////////////////////////

  std::vector<operation> operation_chain;
  // z1 update
  oc_push_back(&operation_chain, false, 1, ndims, "Z1", "Z2", "Xhat", opt);
  oc_push_back(&operation_chain, true , 0, ndims, "X", "Xhat", "A", opt);
  oc_push_back(&operation_chain, true , 1, ndims, "M", "A", "A", opt);
  oc_push_back(&operation_chain, false, 1, ndims, "A", "Z2", "D1_z1", opt);
  oc_push_back(&operation_chain, false, 1, ndims, "M", "Z2", "D2_z1", opt);
  oc_push_back(&operation_chain, true , 0, ndims, "D1_z1", "D2_z1", "D1_z1", opt);
  oc_push_back(&operation_chain, true , 1, ndims, "Z1", "D1_z1", "Z1", opt);

  // z2 update
  oc_push_back(&operation_chain, false, 1, ndims, "Z1", "Z2", "Xhat", opt);
  oc_push_back(&operation_chain, true , 0, ndims, "X", "Xhat", "A", opt);
  oc_push_back(&operation_chain, true , 1, ndims, "M", "A", "A", opt);
  oc_push_back(&operation_chain, false, 1, ndims, "A", "Z1", "D1_z2", opt);
  oc_push_back(&operation_chain, false, 1, ndims, "M", "Z1", "D2_z2", opt);
  oc_push_back(&operation_chain, true , 0, ndims, "D1_z2", "D2_z2", "D1_z2", opt);
  oc_push_back(&operation_chain, true , 1, ndims, "Z2", "D1_z2", "Z2", opt);


  for (int iter=0; iter<30; iter++){
    //if (opt == nmf_gpu)

      operate(&operation_chain);

    /*    else{
      ///////////////////////////////////////////////////////////////////// siil ////
      std::vector<operation>::iterator it;
      for ( it=operation_chain.begin() ; it < operation_chain.end(); it++ ){

        size_t h_zero_cardinality_dim_tuple_size_C = 0;
        size_t h_zero_cardinality_dim_tuples_C_element_number = 0;
        size_t* h_zero_cardinality_dim_tuples_C = NULL;

        if ( it->isHadamard == false){
          std::vector<size_t> zero_cardinality_dims;
          //std::vector<size_t> non_zero_cardinality_dims;
          for ( size_t dim=0; dim<ndims; dim++ ){
            if ( h_objs[it->C]->cardinalities[dim] == 0 && h_objs[it->F]->cardinalities[dim] != 0 ){
              zero_cardinality_dims.push_back(h_objs[it->F]->cardinalities[dim]);
            }
            // else{
            //   non_zero_cardinality_dims.push_back(h_objs[it->F]->cardinalities[dim]);
            // }
          }

          // std::cout << "non_zero_cardinality_dims" << std::endl;
          // for ( size_t j=0; j<non_zero_cardinality_dims.size(); j++){
          //   std::cout << non_zero_cardinality_dims.at(j) << std::endl;
          // }

          if ( COUT ) {
            std::cout << "zero_cardinality_dims" << std::endl;
            for ( size_t j=0; j<zero_cardinality_dims.size(); j++){
              std::cout << zero_cardinality_dims.at(j) << std::endl;
            }
          }

          h_zero_cardinality_dim_tuple_size_C = zero_cardinality_dims.size();


          //h_zero_cardinality_dim_tuples_C_element_number; // set by gen_range_permutation

          h_zero_cardinality_dim_tuples_C =
            gen_range_permutation(zero_cardinality_dims,
                                  &(h_zero_cardinality_dim_tuples_C_element_number));
        }

        mct_tensorop_cpp(it->isHadamard, *(h_objs[it->A]), *(h_objs[it->B]), *(h_objs[it->C]), NULL, *(h_objs[it->F]), ndims, h_zero_cardinality_dim_tuples_C_element_number, h_zero_cardinality_dim_tuples_C);
      }
    }
    */

    //////////////////////////////////////////////////////////////////////////////////



    //    if (iter % 10 == 0 || iter==99 || iter == 98){
    // std::cout << "iter " << iter << std::endl;
    // if (opt == nmf_gpu) transferFromDevice(Z1.data, "Z1");
    // print_ct("current Z1", &Z1, true);

    // if (opt == nmf_gpu) transferFromDevice(Z2.data, "Z2");
    // print_ct("current Z2", &Z2, true);
    //}

  }


  ///////////////////////////////////////////////////////////////////////////////////////////

  // transfer results to matlab /////////////////////////////////////////////////////////////

  if ( opt == nmf_gpu){
    transferFromDevice(m_Z1, "Z1");
    transferFromDevice(m_Z2, "Z2");
  }else if ( opt == nmf_cpp){
    memcpy(m_Z1, Z1.data, Z1.mem_size);
    memcpy(m_Z2, Z2.data, Z2.mem_size);
  }

  ///////////////////////////////////////////////////////////////////////////////////////////


  // reset device
  if (opt == nmf_gpu)
    resetDevice();
}






void mexFunction( int nlhs, mxArray *plhs[], int nrhs, const mxArray *prhs[])
{
  if ( COUT ) std::cout << "mct: found " << nrhs << " number of arguments " << std::endl;

  if( nrhs == 0 ){
    std::cout << "mct: not enough input arguments" << std::endl;
    // print help;
    return;
  }

  if (!mxIsChar(prhs[0]) || (mxGetM(prhs[0]) != 1 ) ){
    mexErrMsgTxt("mct: first argument must be a string.");
    // print help;
    return;
  }
  mwSize buflen = mxGetN(prhs[0])*sizeof(mxChar)+1;
  char* op_name = (char*) mxMalloc(buflen);
  op_type opt;
  int status = mxGetString(prhs[0], op_name, buflen);


  if (strcmp(op_name, "nmf_gpu") == 0){
    if ( COUT ) std::cout << "selecting NMF operation on GPU" << std::endl;
    opt=nmf_gpu;
  }else if (strcmp(op_name, "nmf_cpp") == 0){
    if ( COUT ) std::cout << "selecting tensor operation on CPU" << std::endl;
    opt=nmf_cpp;
  }else if (strcmp(op_name, "tensor_gpu") == 0){
    if ( COUT ) std::cout << "selecting tensor operation on GPU" << std::endl;
    opt=tensor_gpu;
  }else if (strcmp(op_name, "tensor_cpp") == 0){
    if ( COUT ) std::cout << "selecting tensor operation on CPU" << std::endl;
    opt=tensor_cpp;
  }else{
    std::cout << "mct: unknown operation: " << op_name << std::endl;
    // print help;
    return;
  }

  if ( opt == nmf_gpu || opt == nmf_cpp ){
    nmfop(nlhs, plhs, nrhs, prhs, opt);
  }else if ( opt == tensor_gpu || opt == tensor_cpp ){
    tensorop(nlhs, plhs, nrhs, prhs, opt);
  }
}
