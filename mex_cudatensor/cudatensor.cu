#include "hip/hip_runtime.h"
/*
 * author: ck
 * 26.04.2011
 * devised from sgemm.cu of Brian Dushaw
 */

#include "mex.h"
#include "hipblas.h"
#include "cutil_inline.h"

#include <iostream>
#include <algorithm>

#include "cuPrintf.cu"

#define BLOCK_SIZE 16

// setup execution parameters
//dim3 threads(BLOCK_SIZE, BLOCK_SIZE);
//dim3 grid(WC / threads.x, HC / threads.y);
int blocks=BLOCK_SIZE;
int threads=128;


// Tensor .* operation. Multiply corresponding entries of tensors A,B of same size
// Store the result in tensor C

// two operators are available
// hadamard product: multiplies each element of input objects elementwise
// C = A .* B
// requires two input tensors A, B as input

// contract product: performs matrix multiplication if elements are 2 dimensional
// C = A * B
// requires five input arguments A, A_cardinalities, B, B_cardinalities, C_cardinalities
// objects (A,B,C) must have same number of dimensions








void print( const mxArray *prhs[], float* output, int total_size);


// cuda tensor operation configuration object
struct ct_config{
  // defines how many dimensions are there
  size_t ndims;

  // defines the maximum possible size of each dimension
  //   for all tensors using this configuration
  // must be allocated dynamically as an array of type size_t
  // size of the array must be equal to ndims
  size_t* cardinalities;

  // total size of the related objects
  // maximum of cardinality of input objects
  // cardinality for an object is found by multiplying object's cardinalities of each dimension
  size_t total_cardinality;

  // number of elements in the data
  size_t element_number;

  // index of the dimension to contract over
  //size_t contract_dim;
};

// cuda tensor object
struct ct{

  // related configuration object
  ct_config* config;

  // defines size of each dimension for this tensor
  // must be allocated dynamically as an array of type size_t
  // size of the array must be equal to config.ndims
  size_t* cardinalities;

  // size of the corresponding data
  size_t mem_size;

  // points to the values of this tensor
  float* data;
};

// compact structure carying pointers to elements of a cudatensor on the device
struct dev_ct_ptrs{
  ct* ct;
  ct_config* ctc;
  size_t* cardinalities;
  float* data;
};


// multiply corresponding elemens of A, B tensors, put result in tensor C
__global__ void
tensorHadamard( ct* C, ct* A, ct* B)
{
  // Block index
  size_t bx = blockIdx.x;
  //int by = blockIdx.y;

  // Thread index
  size_t tx = threadIdx.x;
  //int ty = threadIdx.y;

  size_t threadsPerblock = blockDim.x * blockDim.y * blockDim.z;
  size_t thread_id = bx * threadsPerblock + tx;

  if ( thread_id  < A->config->element_number ){
    C->data[thread_id] = A->data[thread_id] * B->data[thread_id];
  }
}

// multiply corresponding elements and contract along specified dimension
__global__ void
tensorContract( ct* C_full, ct* C, ct* A, ct* B )
{
  size_t thread_id = threadIdx.x + (threadIdx.y * blockDim.x) + (threadIdx.x * threadIdx.y * blockDim.y);
  size_t block_id = blockIdx.x + (blockIdx.y * gridDim.x);

  // assumes same total dimensions and cardinalities for all objects
  size_t tot_card = A->config->total_cardinality;
  size_t ndims = A->config->ndims;

  if ( thread_id  < tot_card && block_id == 0){

    //size_t uclu[3];
    //for (size_t i=0; i<3; i++) {uclu[i]=0; }

    int index_number_A=0;
    int index_number_B=0;
    int index_number_C=0;

    for (size_t obj=0; obj<2; obj++){
      ct* p;
      if      (obj==0)  p = A;
      else if (obj==1)  p = B;
      //else if (obj==2)  p = C;

      size_t t_id_rem = thread_id;
      size_t cumulative_offset_ind = 1;
      size_t cumulative_offset_elnum = 1;
      size_t cur_card_index=0;
      for (size_t card_index=0; card_index < ndims; card_index++){
        if ( t_id_rem == 0 ) break;

        //uclu[card_index] = (t_id_rem % p->config->cardinalities[card_index]) * cumulative_offset_ind;
        cur_card_index = (t_id_rem % p->config->cardinalities[card_index]) * cumulative_offset_ind;

        t_id_rem = (size_t) t_id_rem / p->config->cardinalities[card_index];

        if (p->cardinalities[card_index] != 0){
	  // int olmazsa patliyor?
	  if      (obj==0)  index_number_A += (int)cur_card_index * (int)cumulative_offset_elnum; 
	  else if (obj==1)  index_number_B += (int)cur_card_index * (int)cumulative_offset_elnum;
	  //else if (obj==2)  index_number_C += (int)cur_card_index * (int)cumulative_offset_elnum;

          // increment cumulative offset with current dimension cardinality for next loop
          // -1 for cardinalities are indexed from 1
          cumulative_offset_ind *= p->config->cardinalities[card_index] - 1 ;
          cumulative_offset_elnum *= p->config->cardinalities[card_index] ;
        }
      }
    }

    size_t tmpB = B->data[index_number_B];
    size_t tmpA= A->data[index_number_A];
    size_t tmpC_full= C_full->data[thread_id];
    cuPrintf("C[%d] %d += A[%d] %d * B[%d] %d\n",  thread_id, tmpC_full,  index_number_A, tmpA, index_number_B, tmpB);

    //~/arastir/cuda2/cudainstall/3.2/sdk/C/src/reduction/doc/reduction.pdf
    // extern __shared__ int sdata[];
    // // each thread loads one element from global to shared mem
    // unsigned int tid = threadIdx.x;
    // unsigned int i = blockIdx.x*blockDim.x + threadIdx.x;
    // sdata[tid] = C->data[i];
    // __syncthreads();

    // sdata[index_number_C] += A->data[index_number_A] * B->data[index_number_B];



    //cuPrintf("C_full->data[%d] = %d ", thread_id, tmpA * tmpB);
    C_full->data[thread_id] = A->data[index_number_A] * B->data[index_number_B];


    __syncthreads();

    // contract on dimensions with zero cardinality
    size_t cum_card=1;
    for (size_t card_index=0; card_index<ndims; card_index++){
      size_t current_card=C->cardinalities[card_index];

      if( current_card == 0 ) {
	// contract on this dimension

	size_t C_ind=0;
	for (size_t C_full_ind=0; C_full_ind < tot_card-1;){

	  size_t tmp1 = C->data[C_ind];
	  size_t tmp2 = C_full->data[C_full_ind];
	  size_t tmp3 = C_full->data[C_full_ind + cum_card];
	  cuPrintf("C[%d] %d +=  C_full[%d] %d + C_full[%d] %d \n", C_ind, tmp1, C_full_ind, tmp2 , C_full_ind+cum_card , tmp3);

	  C->data[C_ind] = C_full->data[C_full_ind] + C_full->data[C_full_ind+cum_card];

	  C_ind++;
	  if (C_full_ind % cum_card == (cum_card-1) ){
	    C_full_ind += cum_card+1;
	  }else{
	    C_full_ind++;
	  }
	}

	// size_t C_full_ind=0;
	// for (size_t C_elnum=0; C_elnum< tot_card/2; C_elnum++){
	//   C->data[C_elnum] = C_full->data[C_full_ind] + C_full->data[C_full_ind+cum_card];
	//   C_full_ind += cum_card;
	//}
      }

      cum_card *= current_card;
    }


    // size_t tmpS= sdata[tid];
    // cuPrintf("C %d\n",tmpS);

    //tmpC= C->data[index_number_C];
    //cuPrintf("C %d\n",tmpC);
    //cuPrintf("uclu %d %d %d index_number %d\n", uclu[0], uclu[1], uclu[2], (int)index_number);
    //cuPrintf("A %d B %d C %d\n", index_number_A, index_number_B, index_number_C);
  }
}


__global__ void
assignCudatensorConfig( ct_config* ctc, size_t* cards ){
  ctc->cardinalities = cards;
}

__global__ void
assignCudatensor( ct* c, ct_config* ctc, size_t* cards, size_t mem_size, float* data){
  c->config = ctc;
  c->cardinalities = cards;
  c->mem_size = mem_size;
  c->data = data;
}



void print_ct_config(char* txt, ct_config* ctc){
  std::cout << txt << std::endl;

  std::cout << "Number of dimensions " << (int) (ctc->ndims) << std::endl;

  //std::cout << "Contract dimension " << (int) (ctc->contract_dim) << std::endl;

  std::cout << "Cardinalities for each dimension of this configuration " << std::endl;
  size_t i=0;
  for ( i=0; i< ctc->ndims; i++){
    std::cout << ctc->cardinalities[i] << " ";
  }
  std::cout << "\nTotal cardinality: " << ctc->total_cardinality << std::endl;
  std::cout << "\nElement number: " << ctc->element_number << std::endl;
  std::cout << std::endl << std::endl << std::endl;
}


void print_ct(char* txt, ct* ct, bool print_config=false, bool printdata=false){

  std::cout << txt << std::endl;

  if (print_config) print_ct_config(txt, ct->config);

  std::cout << "Mem size " << ct->mem_size << std::endl;

  std::cout << "Cardinalities for each dimension of this object "<< std::endl;
  for (size_t i=0; i< ct->config->ndims; i++){
    std::cout << ct->cardinalities[i] << " ";
  }
  std::cout << std::endl;

  if (printdata){
    std::cout << "Data" << std::endl;
    for (size_t i=0; i< ct->config->element_number; i++){
      std::cout << ct->data[i] << " ";
    }
  }
  std::cout << std::endl << std::endl << std::endl;
}

// returns a dev_ct_ptrs struct with information about the cudatensor generated on the device
dev_ct_ptrs prepareDeviceTensor(ct_config* h_ctc, ct_config* d_ctc, ct* h_ct,
                                const mxArray* data, const mxArray* tensor_card = NULL){

 // generate h_ct

  h_ct->config = h_ctc;
  h_ct->cardinalities = (size_t*) malloc(sizeof(size_t)*h_ctc->ndims);

  // assign cardinalities for the tensor objects
  const mwSize* dims_c = mxGetDimensions(data);
  for (size_t i=0; i<h_ctc->ndims; i++){
    if (tensor_card==NULL){
      // we are doing hadamard multiplication, all tensors have same cardinalities
      // or we are doing output tensor object, which as maximum cardinalities on all dimensions
      h_ct->cardinalities[i] = dims_c[i];
      std::cout << "H dim "<< i << " cardinality assignment: "
                << h_ct->cardinalities[i]
                << " <- " << dims_c[i]
                << std::endl;
    }else{
      // we are doing tensor contraction, tensors may have different cardinalities
      h_ct->cardinalities[i] = ((float *)mxGetData(tensor_card))[i];
      std::cout << "TC dim "<< i << " cardinality assignment: "
                << h_ct->cardinalities[i]
                << " <- " << ((float *)mxGetData(tensor_card))[i] << std::endl;
    }
  }


  // assign h_ct host data
  size_t elnum = (size_t) mxGetNumberOfElements(data);
  std::cout << " prepareDeviceTensor elnum " << elnum << std::endl;
  h_ct->mem_size= sizeof(float) * elnum;
  h_ct->data = (float*)malloc( h_ct->mem_size );
  memcpy(h_ct->data, (float*)mxGetData(data), h_ct->mem_size);

  print_ct("prepareDeviceTensor h_ct",h_ct,false,true);


  // allocate d_ct
  ct* d_ct;
  cutilSafeCall(hipMalloc((void**) &d_ct, sizeof(ct)));

  // allocate d_ct contents
  // config -> d_ctc
  size_t* tmp_card;
  cutilSafeCall(hipMalloc((void**)&tmp_card, sizeof(size_t)*h_ctc->ndims));
  cutilSafeCall(hipMemcpy(tmp_card, h_ct->cardinalities, sizeof(size_t)*h_ctc->ndims  ,hipMemcpyHostToDevice));

  float* tmp_data;
  cutilSafeCall(hipMalloc((void**)&tmp_data, h_ct->mem_size));
  cutilSafeCall(hipMemcpy(tmp_data, h_ct->data, h_ct->mem_size, hipMemcpyHostToDevice));

  // put contents of d_ct in their places on the device
  assignCudatensor<<<1, 1>>>(d_ct, d_ctc, tmp_card, h_ct->mem_size, tmp_data);


  dev_ct_ptrs dcp;
  dcp.ct=d_ct;
  dcp.ctc=d_ctc;
  dcp.cardinalities=tmp_card;
  dcp.data=tmp_data;
  return dcp;
}

ct_config* ctcToDevice(ct_config* h_ctc){
  // transfer to device
  size_t* tmp_card;
  cutilSafeCall(hipMalloc((void**)&tmp_card, sizeof(size_t)*h_ctc->ndims));
  cutilSafeCall(hipMemcpy(tmp_card, h_ctc->cardinalities, sizeof(size_t)*h_ctc->ndims ,hipMemcpyHostToDevice));

  ct_config* d_ctc;
  cutilSafeCall(hipMalloc((void**) &d_ctc, sizeof(ct_config) ));
  cutilSafeCall(hipMemcpy( d_ctc , h_ctc, sizeof(ct_config), hipMemcpyHostToDevice) );

  assignCudatensorConfig<<<1,1>>>(d_ctc, tmp_card);

  return d_ctc;
}

ct_config* prepareDeviceTensorConfig(ct_config* h_ctc, const mxArray* sampleObject){
  h_ctc->ndims = mxGetNumberOfDimensions(sampleObject);
  h_ctc->cardinalities = (size_t*) malloc(sizeof(size_t)*h_ctc->ndims);
  const mwSize *dims = mxGetDimensions(sampleObject);
  h_ctc->total_cardinality = 1;
  for (size_t i=0; i<h_ctc->ndims; i++){
    h_ctc->cardinalities[i] = dims[i];
    if(dims[i] != 0)
      h_ctc->total_cardinality *= dims[i];
  }
  return ctcToDevice(h_ctc);
}

ct_config* getDeviceTensorContractConfig(ct_config* h_ctc, const mxArray* tensor1, const mxArray* tensor1_card, const mxArray* tensor2, const mxArray* tensor2_card){
  h_ctc->ndims = mxGetNumberOfElements(tensor1_card); // assumes both objects of same size
  h_ctc->cardinalities = (size_t*) malloc(sizeof(size_t)*h_ctc->ndims);
  h_ctc->element_number = 0;
  h_ctc->total_cardinality = 1;

  float tmpcard1[h_ctc->ndims];
  float tmptotalcard1=1;

  float tmpcard2[h_ctc->ndims];
  float tmptotalcard2=1;

  for (size_t i=0; i<h_ctc->ndims; i++){
    // assumes same total cardinality for all objects
    if ( ((float*)mxGetData(tensor1_card))[i] != 0 )
      h_ctc->total_cardinality *= ((float*)mxGetData(tensor1_card))[i];
    else if (((float*)mxGetData(tensor2_card))[i] != 0)
      h_ctc->total_cardinality *= ((float*)mxGetData(tensor2_card))[i];

    tmpcard1[i] = ((float*)mxGetData(tensor1_card))[i];
    if ( ((float*)mxGetData(tensor1_card))[i] != 0 )
      tmptotalcard1 *= ((float*)mxGetData(tensor1_card))[i];

    tmpcard2[i] = ((float*)mxGetData(tensor2_card))[i];
    if ( ((float*)mxGetData(tensor2_card))[i] != 0 )
      tmptotalcard2 *= ((float*)mxGetData(tensor2_card))[i];
  }

  if (tmptotalcard1 != tmptotalcard2){
    std::cout << "input arguments have different number of elements, exiting" << std::endl;
  }
  std::cout << "element number <- " << tmptotalcard1 << std::endl;
  h_ctc->element_number = tmptotalcard1;

  for (size_t i=0; i<h_ctc->ndims; i++){
    h_ctc->cardinalities[i] = std::max( ((float*)mxGetData(tensor1_card))[i] ,
                                        ((float*)mxGetData(tensor2_card))[i] );
  }

  return ctcToDevice(h_ctc);
}

void print_device_ctc(char* txt, ct_config* d_ctc){
  ct_config tmp_ctc;
  cutilSafeCall(hipMemcpy(&tmp_ctc, d_ctc, sizeof(ct_config), hipMemcpyDeviceToHost));
  //print_ct_config(txt,&tmp_ctc); // must return pointer set from config copy operation and use that
}

void print_device_ct(char* txt,dev_ct_ptrs* dcp, ct* host_ct){
  ct tmp_ct;
  cutilSafeCall(hipMemcpy(&tmp_ct, dcp->ct, sizeof(ct), hipMemcpyDeviceToHost));

  tmp_ct.config = (ct_config*) malloc( sizeof(ct_config) );
  tmp_ct.cardinalities = (size_t*) malloc( host_ct->config->ndims );
  tmp_ct.data = (float*) malloc(host_ct->mem_size);

  cutilSafeCall(hipMemcpy(tmp_ct.data, dcp->data, host_ct->mem_size, hipMemcpyDeviceToHost));
  cutilSafeCall(hipMemcpy(tmp_ct.config, dcp->ctc, sizeof(ct_config), hipMemcpyDeviceToHost));
  cutilSafeCall(hipMemcpy(tmp_ct.cardinalities, dcp->cardinalities, sizeof(size_t)*host_ct->config->ndims, hipMemcpyDeviceToHost));

  print_ct(txt,&tmp_ct,false,true);
}


enum tensor_operation{
  hadamard,
  contract
};


void operate(ct_config* h_ctc, ct_config* d_ctc, const mxArray *prhs[], mxArray *plhs[], tensor_operation operation){

  // input tensor A
  ct h_it_A; dev_ct_ptrs d_A;
  // input tensor B
  ct h_it_B; dev_ct_ptrs d_B;

  if (operation==hadamard){
    // we are doing hadamard multiplication, all tensors have same cardinalities
    std::cout << "d_A prepareDeviceTensor " << std::endl;
    d_A=prepareDeviceTensor(h_ctc, d_ctc, &h_it_A, prhs[0]);
    std::cout << "d_B prepareDeviceTensor " << std::endl;
    d_B=prepareDeviceTensor(h_ctc, d_ctc, &h_it_B, prhs[1]);
  }else if (operation==contract){
    // we are doing tensor contraction, tensors may have different cardinalities
    std::cout << "d_A prepareDeviceTensor " << std::endl;
    d_A=prepareDeviceTensor(h_ctc, d_ctc, &h_it_A, prhs[0], prhs[1]);
    std::cout << "d_B prepareDeviceTensor " << std::endl;
    d_B=prepareDeviceTensor(h_ctc, d_ctc, &h_it_B, prhs[2], prhs[3]);
  }

  // output tensor C
  ct h_ot_C;
  ct h_ot_C_full;
  //ct_config h_ctc_full;
  //ct_config d_ctc_full;

  dev_ct_ptrs d_C;
  dev_ct_ptrs d_C_full;
  mxArray* full_data = mxCreateNumericArray(h_ctc->ndims,h_ctc->cardinalities,mxSINGLE_CLASS,mxREAL);
  mxArray* full_cardinalities = mxCreateNumericArray(h_ctc->ndims,h_ctc->cardinalities,mxSINGLE_CLASS,mxREAL);
  float* f_c_ptr = (float*)mxGetData(full_cardinalities);
  for(size_t i=0; i<h_ctc->ndims; i++ ){
    f_c_ptr[i]=h_ctc->cardinalities[i];
  }


  // prepare MATLAB storage
  float* m_C;
  //size_t m_C_mem_size=1;
  // calculate total cardinalities for all objects
  if(operation == hadamard){
    mwSize argMatDims[h_ctc->ndims];
    for (size_t i=0; i<h_ctc->ndims; i++){
      argMatDims[i] = h_ctc->cardinalities[i];
    }
    plhs[0] = mxCreateNumericArray(h_ctc->ndims,argMatDims,mxSINGLE_CLASS,mxREAL);
    m_C = (float*) mxGetPr(plhs[0]);
    std::cout << "d_C prepareDeviceTensor " << std::endl;
    d_C=prepareDeviceTensor(h_ctc, d_ctc, &h_ot_C, plhs[0]);
  }

  else if (operation == contract){
    size_t non_zero_dim_number=0;
    for (size_t i=0; i<h_ctc->ndims; i++){
      //std::cout << " non_zero_dim_number loop " << i ;
      float tmpdimcard = ((float*)mxGetData(prhs[4]))[i];
      if(tmpdimcard != 0) {
        non_zero_dim_number++;
        //std::cout  << " tmpdimcard " << tmpdimcard << std::endl;
        //m_C_mem_size *= tmpdimcard;
      }
    }

    mwSize argMatDims[non_zero_dim_number];
    size_t argMatDims_ind=0;
    //std::cout << "C tensor init argMatDims with size " << non_zero_dim_number << std::endl;
    //<< " m_C_mem_size " << m_C_mem_size << std::endl;

    for (size_t i=0; i<h_ctc->ndims; i++){
      float val=((float*)mxGetData(prhs[4]))[i];
      //std::cout << "C tensor argMatDims[" << i << "] = " << val << " ";
      if ( val != 0){ // skip dimensions with 0 cardinality
        //std::cout << " assign " << std::endl;
        argMatDims[argMatDims_ind] = val;
        argMatDims_ind++;
      }else{
        //std::cout << " not assign " << std::endl;
      }
    }

    plhs[0] = mxCreateNumericArray(non_zero_dim_number,argMatDims,mxSINGLE_CLASS,mxREAL);
    //std::cout << "SELAM  " <<  (size_t) mxGetNumberOfElements(plhs[0]) << std::endl;
    m_C = (float*) mxGetPr(plhs[0]);

    std::cout << "d_C prepareDeviceTensor " << std::endl;
    d_C=prepareDeviceTensor(h_ctc, d_ctc, &h_ot_C, plhs[0], prhs[4]);

    std::cout << "SELAAM bu " << h_ctc->element_number << std::endl;
    std::cout << "d_C_full prepareDeviceTensor " << std::endl;
    d_C_full=prepareDeviceTensor(h_ctc, d_ctc, &h_ot_C_full, full_data, full_cardinalities);
  }


  bool printdata=true;
  print_ct("Host A",&h_it_A,false,printdata);
  print_ct("Host B",&h_it_B,false,printdata);
  print_ct("Host C",&h_ot_C,false,printdata);
  print_ct("Host C_full",&h_ot_C_full,false,printdata);


  print_device_ct("Device A",&d_A, &h_it_A);
  print_device_ct("Device B",&d_B, &h_it_B);
  print_device_ct("Device C",&d_C, &h_ot_C);

  print_device_ct("Device C_full",&d_C_full, &h_ot_C_full);


  cudaPrintfInit();

  // allocate device memory for result
  // kernel warmup

  // if (operation == hadamard){
  //   tensorHadamard<<< blocks, threads >>>(d_C.ct, d_A.ct, d_B.ct);
  // }else if (operation == contract){
  //   tensorContract<<< blocks, threads >>>(d_C.ct, d_A.ct, d_B.ct);
  // }



  hipDeviceSynchronize();

  // create and start timer
  std::cout << "Run Kernels...\n\n" << std::endl;

  unsigned int timer = 0;
  cutilCheckError(cutCreateTimer(&timer));
  cutilCheckError(cutStartTimer(timer));

  // execute the kernel
  int nIter = 30;
  //for (int j = 0; j < nIter; j++) {
  if (operation == hadamard){
    tensorHadamard<<< blocks, threads >>>(d_C.ct, d_A.ct, d_B.ct);
  }else if (operation == contract){
    tensorContract<<< blocks, threads >>>(d_C_full.ct, d_C.ct, d_A.ct, d_B.ct);
  }
  //}

  // check if kernel execution generated and error
  cutilCheckMsg("Kernel execution failed");

  hipDeviceSynchronize();
  // stop and destroy timer
  cutilCheckError(cutStopTimer(timer));
  //double dSeconds = cutGetTimerValue(timer)/((double)nIter * 1000.0);
  //double dNumOps = 2.0 * total_size;
  //double gflops = 1.0e-9 * dNumOps/dSeconds;
  //std::cout << "tensorMul, Throughput = "<< gflops  << " GFlop/s, Time = " << dSeconds << " s, Size = " << dNumOps <<  " Ops, NumDevsUsed = 1, Workgroup = " << threads << "\n" ;

  cutilCheckError(cutDeleteTimer(timer));

  // copy result from device to host
  float h_C_full_data[h_ctc->total_cardinality];

  if(operation==hadamard){
    cutilSafeCall(hipMemcpy(m_C, d_C.data, h_ot_C.mem_size, hipMemcpyDeviceToHost) ); // assumes same size
  }
  else if(operation==contract){
    cutilSafeCall(hipMemcpy(m_C, d_C.data, h_ot_C.mem_size, hipMemcpyDeviceToHost) ); // assumes same size
    cutilSafeCall(hipMemcpy(h_C_full_data, d_C_full.data, h_ot_C_full.mem_size, hipMemcpyDeviceToHost) );
  }

  // print C_full 
  for (size_t i=0; i<h_ctc->total_cardinality; i++){
    std::cout << "C_full[" << i << "] = " << h_C_full_data[i] << std::endl;
  }


  // clean up memory
  //free(h_A);
  //free(h_B);
  //free(h_C);
  //free(reference);

  // wrong
  //cutilSafeCall(hipFree());
  //cutilSafeCall(hipFree(d_it_B));
  //cutilSafeCall(hipFree(d_it_A)); //->C

  print_device_ct("Result\nDevice C",&d_C_full, &h_ot_C);
  print_device_ct("Result\nDevice C",&d_C, &h_ot_C);

  cudaPrintfDisplay(stdout, true);
  cudaPrintfEnd();

  hipDeviceReset();


  // required to avoid memory leak?
  delete h_ctc->cardinalities;
  delete h_it_A.cardinalities;
  delete h_it_B.cardinalities;
}



void mexFunction( int nlhs, mxArray *plhs[], int nrhs, const mxArray *prhs[])
{

  std::cout << "mex: found " << nrhs << " number of arguments " << std::endl;
  if (nrhs == 2){
    // hadamard multiplication
    std::cout << "mex: applying hadamard multiplication " << std::endl;

    // ASSUMES target tensors are of the same dimension
    ct_config h_ctc;
    ct_config* d_ctc = prepareDeviceTensorConfig(&h_ctc,prhs[0]);

    print_ct_config("Host ctc",&h_ctc);

    operate(&h_ctc, d_ctc, prhs, plhs, hadamard);

  }else if(nrhs==5){
    // tensor contraction operation
    std::cout << "mex: applying tensor contraction " << std::endl;

    ct_config h_ctc;
    ct_config* d_ctc = getDeviceTensorContractConfig(&h_ctc,prhs[0],prhs[1],prhs[2],prhs[3]);

    print_ct_config("Host ctc", &h_ctc);

    print_device_ctc("Device tmp ctc",d_ctc);

    operate(&h_ctc, d_ctc, prhs, plhs, contract);


  }else{
    std::cout << "mex: wrong number of arguments " << std::endl;
  }


}


void print( const mxArray *prhs[], float* output, int total_size){
  std::cout << "\ntotal_size " << total_size << std::endl;

  std::cout << std::endl << std::endl << "input A:" << std::endl;
  for (int i=0; i<total_size ; i++){
    std::cout << i << "\t" << ((float*)mxGetPr(prhs[0]))[i] << std::endl;
  }

  std::cout << std::endl << std::endl << "input B:" << std::endl;
  for (int i=0; i<total_size ; i++){
    std::cout << i << "\t" << ((float*)mxGetPr(prhs[1]))[i]  << std::endl;
  }

  std::cout << std::endl << std::endl << "output C:" << std::endl;
  for (int i=0; i<total_size ; i++){
    std::cout << i << "\t" <<  output[i]  << std::endl;
  }
}
